#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 5
#define TW 3
#define TC 4
#define C 64
#define N 32
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[5184];
  __shared__ float kernel_shared[4608];
  float pad_temp_shared_local[4];
  float kernel_shared_local[16];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  pad_temp_shared[((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)))] = (((((9 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 81)) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) / 81) * 49)) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 1))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 2))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 3))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 4))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 5))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 6))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 7))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 8))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 9))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 9) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 9) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 9) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 9) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 10))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 10) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 10) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 10) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 10) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 11))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 11) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 11) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 11) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 11) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 12))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 12) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 12) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 12) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 12) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 13))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 13) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 13) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 13) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 13) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 14))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 14) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 14) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 14) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 14) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 15))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 15) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 15) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 15) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 15) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 16))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 16) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 16) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 16) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 16) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 17))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 17) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 17) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 17) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 17) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 18))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 18) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 18) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 18) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 18) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 19))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 19) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 19) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 19) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 19) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 20))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 20) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 20) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 20) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 20) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 21))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 21) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 21) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 21) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 21) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 22))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 22) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 22) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 22) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 22) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 23))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 23) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 23) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 23) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 23) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 24))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 24) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 24) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 24) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 24) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 25))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 25) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 25) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 25) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 25) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 26))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 26) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 26) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 26) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 26) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 27))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 27) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 27) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 27) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 27) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 28))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 28) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 28) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 28) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 28) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 29))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 29) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 29) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 29) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 29) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 30))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 30) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 30) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 30) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 30) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 31))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 31) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 31) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 31) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 31) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 32))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 32) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 32) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 32) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 32) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 33))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 33) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 33) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 33) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 33) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 34))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 34) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 34) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 34) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 34) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 35))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 35) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 35) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 35) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 35) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 36))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 36) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 36) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 36) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 36) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 37))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 37) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 37) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 37) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 37) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 38))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 38) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 38) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 38) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 38) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 39))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 39) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 39) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 39) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 39) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 40))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 40) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 40) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 40) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 40) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 41))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 41) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 41) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 41) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 41) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 42))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 42) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 42) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 42) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 42) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 43))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 43) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 43) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 43) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 43) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 44))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 44) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 44) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 44) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 44) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 8) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 45))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 45) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 45) % 81) < 72)) && (1 <= (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9))) && ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 45) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 45) % 81) / 9) * 7)) + (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 46))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 46) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 46) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 46) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 46) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 1) % 9)) - 8))] : 0.000000e+00f);
  pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 47))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 47) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 47) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 47) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 47) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 2) % 9)) - 8))] : 0.000000e+00f);
  if (((((int)threadIdx.z) * 32) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) / 81)) < 64) {
    if (((((int)threadIdx.z) * 288) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) / 9)) < 576) {
      if ((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) < 5136) {
        if (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) < 2544) {
          pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 48))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 48) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 3) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 32) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) / 81)) < 64) {
    if (((((int)threadIdx.z) * 288) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) / 9)) < 576) {
      if ((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) < 5135) {
        if (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) < 2543) {
          pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 49))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 49) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 4) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 32) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) / 81)) < 64) {
    if (((((int)threadIdx.z) * 288) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) / 9)) < 576) {
      if ((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) < 5134) {
        if (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) < 2542) {
          pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 50))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 50) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 5) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 32) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) / 81)) < 64) {
    if (((((int)threadIdx.z) * 288) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) / 9)) < 576) {
      if ((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) < 5133) {
        if (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) < 2541) {
          pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 51))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 51) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 6) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 32) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) / 81)) < 64) {
    if (((((int)threadIdx.z) * 288) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) / 9)) < 576) {
      if ((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) < 5132) {
        if (((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) < 2540) {
          pad_temp_shared[(((((((int)threadIdx.z) * 2592) + (((int)threadIdx.y) * 371)) + (((int)threadIdx.x) * 53)) + 52))] = (((((9 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) % 81)) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9))) && (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9) < 8)) ? data[((((((((int)threadIdx.z) * 1568) + (((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) / 81) * 49)) + ((((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 52) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 371) + (((int)threadIdx.x) * 53)) + 7) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)))] = kernel[(((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 1))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 1))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 2))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 2))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 3))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 3))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 4))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 4))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 5))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 5))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 6))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 6))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 7))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 7))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 8))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 8))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 9))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 9))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 10))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 10))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 11))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 11))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 12))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 12))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 13))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 13))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 14))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 14))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 15))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 15))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 16))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 16))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 17))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 17))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 18))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 18))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 19))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 19))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 20))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 20))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 21))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 21))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 22))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 22))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 23))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 23))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 24))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 24))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 25))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 25))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 26))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 26))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 27))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 27))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 28))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 28))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 29))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 29))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 30))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 30))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 31))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 31))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 32))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 32))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 33))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 33))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 34))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 34))];
  kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 35))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 35))];
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 508) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1524) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4572) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2268) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 36))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 36))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 508) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1524) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4571) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2267) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 37))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 37))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 508) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1524) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4570) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2266) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 12) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 38))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 38))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1523) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4569) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2265) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 39))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 39))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1523) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4568) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2264) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 40))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 40))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1523) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4567) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2263) {
            if ((((((int)blockIdx.z) * 8) + (((int)threadIdx.z) * 4)) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 13) / 192)) < 32) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 41))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 41))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1522) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4566) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2262) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 42))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 42))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1522) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4565) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2261) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 43))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 43))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 14) / 3)) < 512) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1522) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4564) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2260) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 44))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 44))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 15) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 507) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1521) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4563) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2259) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 45))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 45))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 15) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 507) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1521) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4562) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2258) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 46))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 46))];
            }
          }
        }
      }
    }
  }
  if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) + 15) / 192)) < 8) {
    if (((((int)threadIdx.z) * 256) + (((((int)threadIdx.y) * 110) + (((int)threadIdx.x) * 16)) / 3)) < 507) {
      if ((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 110)) + (((int)threadIdx.x) * 16)) < 1521) {
        if ((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) < 4561) {
          if (((((int)threadIdx.y) * 330) + (((int)threadIdx.x) * 48)) < 2257) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 2304) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 47))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 330)) + (((int)threadIdx.x) * 48)) + 47))];
            }
          }
        }
      }
    }
  }
  __syncthreads();
  for (int rc_inner_outer = 0; rc_inner_outer < 16; ++rc_inner_outer) {
    pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 81))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 162))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 243))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1152))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2304))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3456))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 9))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1161))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2313))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3465))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 18))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1170))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2322))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3474))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 27))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1179))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2331))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3483))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 82))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 163))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 244))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1153))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2305))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3457))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 10))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1162))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2314))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3466))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 19))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1171))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2323))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3475))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 28))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1180))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2332))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3484))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 83))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 164))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 245))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1154))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2306))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3458))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 11))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1163))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2315))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3467))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 20))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1172))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2324))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3476))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 29))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1181))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2333))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3485))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 90))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 171))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 252))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1155))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2307))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3459))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 12))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1164))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2316))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3468))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 21))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1173))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2325))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3477))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 30))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1182))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2334))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3486))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 91))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 172))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 253))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 4))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1156))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2308))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3460))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 13))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1165))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2317))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3469))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 22))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1174))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2326))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3478))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 31))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1183))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2335))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3487))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 11))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 92))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 173))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 254))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 5))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1157))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2309))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3461))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 14))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1166))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2318))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3470))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 23))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1175))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2327))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3479))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 32))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1184))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2336))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3488))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 99))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 180))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 261))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 6))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1158))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2310))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3462))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 15))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1167))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2319))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3471))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 24))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1176))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2328))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3480))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 33))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1185))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2337))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3489))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 19))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 100))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 181))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 262))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 7))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1159))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2311))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3463))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 16))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1168))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2320))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3472))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 25))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1177))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2329))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3481))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 34))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1186))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2338))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3490))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 101))];
    pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 182))];
    pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 324) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 263))];
    kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 8))];
    kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1160))];
    kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2312))];
    kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3464))];
    kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 17))];
    kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1169))];
    kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2321))];
    kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3473))];
    kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 26))];
    kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1178))];
    kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2330))];
    kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3482))];
    kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 35))];
    kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 1187))];
    kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 2339))];
    kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 36)) + 3491))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 98))] = compute_local[(1)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(2)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 294))] = compute_local[(3)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*3+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*3+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*3+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*3+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*3+(4-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*3+(0-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*3+(1-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*3+(2-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*3+(3-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*3+(4-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*3+(0-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*3+(1-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*3+(2-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*3+(3-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*3+(4-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*3+(0-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*3+(1-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*3+(2-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*3+(3-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*3+(4-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*3+(0-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*3+(1-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*3+(2-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*3+(3-s)] += result;
				}
			}
		break;
		case 24:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*3+(4-s)] += result;
				}
			}
		break;
		case 25:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*3+(0-s)] += result;
				}
			}
		break;
		case 26:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*3+(1-s)] += result;
				}
			}
		break;
		case 27:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*3+(2-s)] += result;
				}
			}
		break;
		case 28:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*3+(3-s)] += result;
				}
			}
		break;
		case 29:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*3+(4-s)] += result;
				}
			}
		break;
		case 30:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*3+(0-s)] += result;
				}
			}
		break;
		case 31:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*3+(1-s)] += result;
				}
			}
		break;
		case 32:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*3+(2-s)] += result;
				}
			}
		break;
		case 33:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*3+(3-s)] += result;
				}
			}
		break;
		case 34:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*3+(4-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,1,4);

        dim3 block(7,7,2);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;



    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


