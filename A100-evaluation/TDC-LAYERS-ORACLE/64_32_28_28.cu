#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 2
#define TC 16
#define C 64
#define N 32
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[384];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[4];
  float kernel_shared_local[3];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)))] = (((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) && (1 <= ((int)blockIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 1))] = ((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 28))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 2))] = ((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 27))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 3))] = ((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 26))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 4))] = ((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 25))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 5))] = (((1 <= (((int)blockIdx.y) + ((int)threadIdx.x))) && (((int)blockIdx.x) < 6)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 24))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 6))] = ((((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) && (1 <= ((int)blockIdx.x))) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) - 1))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 7))] = (((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) ? data[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 8))] = (((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) + 1))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 9))] = (((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) + 2))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 10))] = (((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) + 3))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 11))] = ((((((int)blockIdx.y) + ((int)threadIdx.x)) < 14) && (((int)blockIdx.x) < 6)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 56)) + (((int)blockIdx.x) * 4)) + 4))] : 0.000000e+00f);
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 1))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 1))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 2))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 2))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 3))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 3))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 4))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 4))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 5))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 5))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 6))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 6))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 7))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 7))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 8))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 8))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 9))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 9))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 10))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 10))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 11))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 11))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 12))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 12))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 13))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 13))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 14))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 14))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 15))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 15))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 16))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 16))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 17))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 17))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 18))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 18))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 19))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 19))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 20))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 20))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 21))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 21))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 22))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 22))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 23))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 23))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 24))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 24))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 25))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 25))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 26))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 26))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 27))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 27))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 28))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 28))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 29))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 29))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 30))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 30))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 31))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 31))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 32))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 32))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 33))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 33))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 34))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 34))];
    kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 35))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 576)) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + 35))];
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 3))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 144))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 6))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 8))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 9))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 3))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 12))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 13))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 15))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 8))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 24))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 25))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 26))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 27))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 9))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 30))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 31))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 33))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 14))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 36))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 37))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 38))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 39))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 15))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 48))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 49))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 50))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 51))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 20))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 54))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 55))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 57))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 21))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 22))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 60))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 61))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 62))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 63))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 26))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 72))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 73))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 74))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 75))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 27))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 28))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 78))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 79))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 80))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 81))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 32))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 84))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 85))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 86))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 87))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 33))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 34))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 99))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 38))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 102))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 103))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 104))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 105))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 39))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 40))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 108))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 109))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 110))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 111))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 44))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 120))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 121))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 122))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 123))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 45))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 46))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 127))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 129))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 50))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 132))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 134))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 135))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 51))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 144))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 145))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 146))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 56))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 150))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 151))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 152))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 153))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 57))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 58))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 156))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 157))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 158))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 159))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 62))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 168))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 169))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 170))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 171))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 63))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 64))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 174))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 175))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 176))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 177))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 68))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 180))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 181))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 182))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 183))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 69))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 70))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 195))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 73))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 74))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 198))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 199))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 200))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 201))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 75))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 76))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 77))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 204))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 205))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 206))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 78))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 79))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 80))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 216))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 217))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 218))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 219))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 81))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 82))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 83))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 222))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 223))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 225))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 84))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 85))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 86))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 228))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 229))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 230))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 231))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 87))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 88))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 89))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 240))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 241))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 242))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 243))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 90))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 91))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 92))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 246))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 247))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 248))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 249))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 93))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 94))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 95))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 253))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 254))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 255))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 96))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 97))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 98))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 264))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 265))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 267))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 99))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 100))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 101))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 270))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 271))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 272))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 273))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 102))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 103))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 104))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 276))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 277))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 278))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 279))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 105))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 106))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 107))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 291))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 108))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 109))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 110))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 294))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 295))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 296))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 297))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 111))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 112))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 113))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 300))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 301))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 302))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 303))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 114))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 115))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 116))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 312))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 313))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 314))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 315))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 117))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 118))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 119))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 318))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 319))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 321))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 120))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 121))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 122))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 324))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 325))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 326))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 327))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 123))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 124))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 125))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 336))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 337))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 338))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 339))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 126))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 127))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 128))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 342))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 343))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 344))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 345))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 129))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 130))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 131))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 348))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 349))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 350))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 351))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 132))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 133))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 134))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 360))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 361))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 362))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 363))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 135))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 136))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 137))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 366))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 367))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 368))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 369))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 138))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 139))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 140))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 372))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 373))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 374))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 2)) + 375))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 141))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 142))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 143))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
  }
  compute[(((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(7,14,4);

                dim3 block(2,2,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


