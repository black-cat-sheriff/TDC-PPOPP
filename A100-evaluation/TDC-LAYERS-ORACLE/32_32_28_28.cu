#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 9
#define TW 2
#define TC 8
#define C 32
#define N 32
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[72];
  float pad_temp_shared_local[24];
  float kernel_shared_local[18];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 28) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 28) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + (((((int)threadIdx.x) * 28) / 96) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 28) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 28) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 1) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 1) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 1) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 1) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 2) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 2) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 2) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 2) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 3) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 3) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 3) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 3) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 4) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 4) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 4) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 4) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 5) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 5) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 5) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 5) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 6) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 6) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 6) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 6) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 7) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 7) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 7) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 7) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 7) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 8) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 8) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 8) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 8) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 8) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 9))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 9) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 9) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 9) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 9) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 9) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 10))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 10) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 10) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 10) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 10) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 10) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 11))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 11) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 11) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 11) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 11) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 11) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 12) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 12) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 12) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 12) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 12) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 12) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 13) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 13) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 13) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 13) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 13) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 13) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 14) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 14) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 14) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 14) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 14) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 14) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 15) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 15) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 15) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 15) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 15) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 15) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 16) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 16) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 16) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 16) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 28) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 17) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 17) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 17) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 17) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 18))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 18) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 18) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 18) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 18) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 19))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 19) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 19) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 19) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 19) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 20))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 20) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 20) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 20) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 20) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 21))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 21) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 21) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 21) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 21) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 22))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 22) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 22) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 22) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 22) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 23))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 23) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 23) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 23) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 23) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 7) & 15)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 24) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 24) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 744) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 24))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 24) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 24) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 24) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 24) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 8) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 25) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 25) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 743) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 25))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 25) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 25) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 25) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 25) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 9) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 26) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 26) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 742) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 26))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 26) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 26) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 26) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 26) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 10) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 27) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 27) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 741) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 27))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 27) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 27) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 27) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 27) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 11) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 72) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[(((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)))] = kernel[(((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 71) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) + 1))] = kernel[((((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 70) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) + 2))] = kernel[((((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 3))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 16))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 17))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 18))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 19))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 33))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 34))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 35))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 99))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 112))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 113))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 114))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 115))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 129))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 130))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 131))];
    kernel_shared_local[(0)] = kernel_shared[(0)];
    kernel_shared_local[(1)] = kernel_shared[(1)];
    kernel_shared_local[(2)] = kernel_shared[(2)];
    kernel_shared_local[(3)] = kernel_shared[(3)];
    kernel_shared_local[(4)] = kernel_shared[(4)];
    kernel_shared_local[(5)] = kernel_shared[(5)];
    kernel_shared_local[(6)] = kernel_shared[(6)];
    kernel_shared_local[(7)] = kernel_shared[(7)];
    kernel_shared_local[(8)] = kernel_shared[(8)];
    kernel_shared_local[(9)] = kernel_shared[(9)];
    kernel_shared_local[(10)] = kernel_shared[(10)];
    kernel_shared_local[(11)] = kernel_shared[(11)];
    kernel_shared_local[(12)] = kernel_shared[(12)];
    kernel_shared_local[(13)] = kernel_shared[(13)];
    kernel_shared_local[(14)] = kernel_shared[(14)];
    kernel_shared_local[(15)] = kernel_shared[(15)];
    kernel_shared_local[(16)] = kernel_shared[(16)];
    kernel_shared_local[(17)] = kernel_shared[(17)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 195))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 208))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 209))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 210))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 211))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 225))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 226))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 227))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 291))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 304))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 305))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 306))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 307))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 321))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 322))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 323))];
    kernel_shared_local[(0)] = kernel_shared[(18)];
    kernel_shared_local[(1)] = kernel_shared[(19)];
    kernel_shared_local[(2)] = kernel_shared[(20)];
    kernel_shared_local[(3)] = kernel_shared[(21)];
    kernel_shared_local[(4)] = kernel_shared[(22)];
    kernel_shared_local[(5)] = kernel_shared[(23)];
    kernel_shared_local[(6)] = kernel_shared[(24)];
    kernel_shared_local[(7)] = kernel_shared[(25)];
    kernel_shared_local[(8)] = kernel_shared[(26)];
    kernel_shared_local[(9)] = kernel_shared[(27)];
    kernel_shared_local[(10)] = kernel_shared[(28)];
    kernel_shared_local[(11)] = kernel_shared[(29)];
    kernel_shared_local[(12)] = kernel_shared[(30)];
    kernel_shared_local[(13)] = kernel_shared[(31)];
    kernel_shared_local[(14)] = kernel_shared[(32)];
    kernel_shared_local[(15)] = kernel_shared[(33)];
    kernel_shared_local[(16)] = kernel_shared[(34)];
    kernel_shared_local[(17)] = kernel_shared[(35)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 386))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 387))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 400))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 401))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 402))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 403))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 416))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 417))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 418))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 419))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 483))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 496))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 497))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 498))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 499))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 512))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 513))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 514))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 515))];
    kernel_shared_local[(0)] = kernel_shared[(36)];
    kernel_shared_local[(1)] = kernel_shared[(37)];
    kernel_shared_local[(2)] = kernel_shared[(38)];
    kernel_shared_local[(3)] = kernel_shared[(39)];
    kernel_shared_local[(4)] = kernel_shared[(40)];
    kernel_shared_local[(5)] = kernel_shared[(41)];
    kernel_shared_local[(6)] = kernel_shared[(42)];
    kernel_shared_local[(7)] = kernel_shared[(43)];
    kernel_shared_local[(8)] = kernel_shared[(44)];
    kernel_shared_local[(9)] = kernel_shared[(45)];
    kernel_shared_local[(10)] = kernel_shared[(46)];
    kernel_shared_local[(11)] = kernel_shared[(47)];
    kernel_shared_local[(12)] = kernel_shared[(48)];
    kernel_shared_local[(13)] = kernel_shared[(49)];
    kernel_shared_local[(14)] = kernel_shared[(50)];
    kernel_shared_local[(15)] = kernel_shared[(51)];
    kernel_shared_local[(16)] = kernel_shared[(52)];
    kernel_shared_local[(17)] = kernel_shared[(53)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 576))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 577))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 578))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 579))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 592))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 593))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 594))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 595))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 608))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 609))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 610))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 611))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 672))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 673))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 674))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 675))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 688))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 689))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 690))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 691))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 704))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 705))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 706))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(54)];
    kernel_shared_local[(1)] = kernel_shared[(55)];
    kernel_shared_local[(2)] = kernel_shared[(56)];
    kernel_shared_local[(3)] = kernel_shared[(57)];
    kernel_shared_local[(4)] = kernel_shared[(58)];
    kernel_shared_local[(5)] = kernel_shared[(59)];
    kernel_shared_local[(6)] = kernel_shared[(60)];
    kernel_shared_local[(7)] = kernel_shared[(61)];
    kernel_shared_local[(8)] = kernel_shared[(62)];
    kernel_shared_local[(9)] = kernel_shared[(63)];
    kernel_shared_local[(10)] = kernel_shared[(64)];
    kernel_shared_local[(11)] = kernel_shared[(65)];
    kernel_shared_local[(12)] = kernel_shared[(66)];
    kernel_shared_local[(13)] = kernel_shared[(67)];
    kernel_shared_local[(14)] = kernel_shared[(68)];
    kernel_shared_local[(15)] = kernel_shared[(69)];
    kernel_shared_local[(16)] = kernel_shared[(70)];
    kernel_shared_local[(17)] = kernel_shared[(71)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
  }
  compute[((((((((int)blockIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(0-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(1-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(2-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(3-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(0-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(1-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(2-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(3-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(0-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(1-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(2-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(3-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(0-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(1-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(2-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*2+(3-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*2+(0-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*2+(1-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*2+(2-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*2+(3-s)] += result;
				}
			}
		break;
		case 24:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*2+(0-s)] += result;
				}
			}
		break;
		case 25:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*2+(1-s)] += result;
				}
			}
		break;
		case 26:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*2+(2-s)] += result;
				}
			}
		break;
		case 27:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(6-r)*2+(3-s)] += result;
				}
			}
		break;
		case 28:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(7-r)*2+(0-s)] += result;
				}
			}
		break;
		case 29:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(7-r)*2+(1-s)] += result;
				}
			}
		break;
		case 30:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(7-r)*2+(2-s)] += result;
				}
			}
		break;
		case 31:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(7-r)*2+(3-s)] += result;
				}
			}
		break;
		case 32:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(8-r)*2+(0-s)] += result;
				}
			}
		break;
		case 33:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(8-r)*2+(1-s)] += result;
				}
			}
		break;
		case 34:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(8-r)*2+(2-s)] += result;
				}
			}
		break;
		case 35:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(8-r)*2+(3-s)] += result;
				}
			}
		break;
		case 36:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(9-r)*2+(0-s)] += result;
				}
			}
		break;
		case 37:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(9-r)*2+(1-s)] += result;
				}
			}
		break;
		case 38:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(9-r)*2+(2-s)] += result;
				}
			}
		break;
		case 39:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(9-r)*2+(3-s)] += result;
				}
			}
		break;
		case 40:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(10-r)*2+(0-s)] += result;
				}
			}
		break;
		case 41:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(10-r)*2+(1-s)] += result;
				}
			}
		break;
		case 42:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(10-r)*2+(2-s)] += result;
				}
			}
		break;
		case 43:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(10-r)*2+(3-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,7,32);

        dim3 block(7,4,1);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;



    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


