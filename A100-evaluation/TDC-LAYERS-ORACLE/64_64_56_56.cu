#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 8
#define TC 16
#define C 64
#define N 64
#define H 56
#define W 56

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[16];
  float kernel_shared_local[24];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + ((((int)threadIdx.x) * 7) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + ((((int)threadIdx.x) * 7) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 7) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 7) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + ((((int)threadIdx.x) * 7) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + ((((int)threadIdx.x) * 7) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 7) & 15)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 1) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 1) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 1) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 1) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 1) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 1) & 15)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 2) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 2) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 2) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 2) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 2) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 2) & 15)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 3) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 3) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 3) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 3) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 3) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 3) & 15)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 4) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 4) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 4) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 4) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 4) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 4) & 15)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 5) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 5) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 5) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 5) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 5) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 5) & 15)) - 57))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 6) >> 4)) / 6)) < 8) {
      if ((((((int)threadIdx.z) * 12) + (((int)threadIdx.y) * 3)) + (((((int)threadIdx.x) * 7) + 6) >> 4)) < 48) {
        if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) < 762) {
          if (((((int)threadIdx.y) * 48) + (((int)threadIdx.x) * 7)) < 186) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 48)) + (((int)threadIdx.x) * 7)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 6) >> 4)) % 6))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 6) >> 4)) % 6)) < 57)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + 6) & 15)) < 57)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 6) >> 4)) / 6) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + 6) >> 4)) % 6) * 56)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + 6) & 15)) - 57))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 192) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 576) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 144) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) & 7) * 9)) + (((((int)threadIdx.x) * 2) % 3) * 3)))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 192) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 575) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 143) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) + 1))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) & 7) * 9)) + (((((int)threadIdx.x) * 2) % 3) * 3)) + 1))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + ((((int)threadIdx.x) * 2) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 192) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 574) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 142) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) + 2))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + ((((int)threadIdx.x) * 2) / 3)) & 7) * 9)) + (((((int)threadIdx.x) * 2) % 3) * 3)) + 2))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 191) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 573) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 141) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) + 3))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) & 7) * 9)) + ((((((int)threadIdx.x) * 2) + 1) % 3) * 3)))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 191) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 572) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 140) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) + 4))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) & 7) * 9)) + ((((((int)threadIdx.x) * 2) + 1) % 3) * 3)) + 1))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3)) < 8) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 4)) + (((((int)threadIdx.x) * 2) + 1) / 3)) < 64) {
        if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 12)) + (((int)threadIdx.x) * 2)) < 191) {
          if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) < 571) {
            if (((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 6)) < 139) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 6)) + 5))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) >> 3) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 2) + 1) / 3)) & 7) * 9)) + ((((((int)threadIdx.x) * 2) + 1) % 3) * 3)) + 2))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 96))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 97))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 98))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 99))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 192))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 193))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 194))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 195))];
      pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 288))];
      pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 289))];
      pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 290))];
      pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 291))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 1))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 2))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 9))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 10))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 11))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 18))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 19))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 20))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 27))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 28))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 29))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 72))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 73))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 74))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 81))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 82))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 83))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 90))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 91))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 92))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 99))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 100))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 101))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(15)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(16)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(17)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(18)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(19)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(20)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(21)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(22)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(23)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(23)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 17))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 18))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 19))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 112))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 113))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 114))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 115))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 208))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 209))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 210))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 211))];
      pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 304))];
      pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 305))];
      pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 306))];
      pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 307))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 3))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 4))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 5))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 12))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 13))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 14))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 21))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 22))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 23))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 30))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 31))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 32))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 75))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 76))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 77))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 84))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 85))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 86))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 93))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 94))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 95))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 102))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 103))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 104))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(15)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(16)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(17)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(18)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(19)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(20)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(21)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(22)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(23)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(23)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 33))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 34))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 35))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 128))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 129))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 130))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 131))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 224))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 225))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 226))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 227))];
      pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 320))];
      pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 321))];
      pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 322))];
      pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 384) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 323))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 6))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 7))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 8))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 15))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 16))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 17))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 24))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 25))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 26))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 33))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 34))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 35))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 78))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 79))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 80))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 87))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 88))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 89))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 96))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 97))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 98))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 105))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 106))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + 107))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(15)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(16)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(17)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(18)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(19)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(20)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(21)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(22)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(23)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(23)]));
    }
  }
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 3136))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 3137))] = compute_local[(3)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 5:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 5; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 6:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 6; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 7:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 7; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 8:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 8; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[0];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[1];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[0];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 8]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 8]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 9]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 8]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 8]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 9]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 8]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 8]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 9]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(4,14,8);

                dim3 block(7,4,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


