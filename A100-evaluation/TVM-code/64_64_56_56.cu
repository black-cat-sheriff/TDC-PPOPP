#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 64
#define N 64
#define H 56
#define W 56

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[2784];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[32];
  float kernel_shared_local[32];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
    compute_local[(ff_c_init)] = 0.000000e+00f;
    compute_local[((ff_c_init + 8))] = 0.000000e+00f;
    compute_local[((ff_c_init + 2))] = 0.000000e+00f;
    compute_local[((ff_c_init + 10))] = 0.000000e+00f;
    compute_local[((ff_c_init + 4))] = 0.000000e+00f;
    compute_local[((ff_c_init + 12))] = 0.000000e+00f;
    compute_local[((ff_c_init + 6))] = 0.000000e+00f;
    compute_local[((ff_c_init + 14))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 348)) < 8) {
        if (((((int)threadIdx.z) * 116) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6)) < 464) {
          if ((((((int)threadIdx.z) * 696) + (((int)threadIdx.y) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 2784) {
            if (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 696) {
              pad_temp_shared[((((((int)threadIdx.z) * 696) + (((int)threadIdx.y) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((6 <= (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348)) && ((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348) < 342)) && (1 <= ((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)))) && (((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) < 57)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 348) * 3136)) + (((((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 348) / 6) * 56)) + (((int)blockIdx.x) * 4)) + (((((int)threadIdx.y) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) - 57))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24)) < 16) {
        if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 3)) < 128) {
          if ((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) < 384) {
            if ((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1152) {
              if (((((int)threadIdx.y) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
                if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 4)) + (((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24)) < 64) {
                  kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 24) * 576)) + (rc_outer * 72)) + ((((((int)threadIdx.y) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) % 24) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
      #pragma unroll
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        #pragma unroll
        for (int ax1 = 0; ax1 < 8; ++ax1) {
          pad_temp_shared_local[(ax1)] = pad_temp_shared[(((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer))];
          pad_temp_shared_local[((ax1 + 8))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 1))];
          pad_temp_shared_local[((ax1 + 16))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 2))];
          pad_temp_shared_local[((ax1 + 24))] = pad_temp_shared[((((((ax1 * 348) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + rx_inner_outer) + 3))];
        }
        #pragma unroll
        for (int ax0 = 0; ax0 < 2; ++ax0) {
          #pragma unroll
          for (int ax11 = 0; ax11 < 8; ++ax11) {
            kernel_shared_local[(((ax0 * 8) + ax11))] = kernel_shared[((((((((int)threadIdx.z) * 144) + (ax0 * 72)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer))];
            kernel_shared_local[((((ax0 * 8) + ax11) + 16))] = kernel_shared[(((((((((int)threadIdx.z) * 144) + (ax0 * 72)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer) + 576))];
          }
        }
        #pragma unroll
        for (int rc_inner_inner = 0; rc_inner_inner < 8; ++rc_inner_inner) {
          #pragma unroll
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[(ff_c)] = (compute_local[(ff_c)] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 8))] = (compute_local[((ff_c + 8))] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 2))] = (compute_local[((ff_c + 2))] + (pad_temp_shared_local[((rc_inner_inner + 8))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 10))] = (compute_local[((ff_c + 10))] + (pad_temp_shared_local[((rc_inner_inner + 8))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 4))] = (compute_local[((ff_c + 4))] + (pad_temp_shared_local[((rc_inner_inner + 16))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 12))] = (compute_local[((ff_c + 12))] + (pad_temp_shared_local[((rc_inner_inner + 16))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
            compute_local[((ff_c + 6))] = (compute_local[((ff_c + 6))] + (pad_temp_shared_local[((rc_inner_inner + 24))] * kernel_shared_local[(((ff_c * 8) + rc_inner_inner))]));
            compute_local[((ff_c + 14))] = (compute_local[((ff_c + 14))] + (pad_temp_shared_local[((rc_inner_inner + 24))] * kernel_shared_local[((((ff_c * 8) + rc_inner_inner) + 16))]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2; ++ff_inner_inner_inner) {
    compute[((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)))] = compute_local[(ff_inner_inner_inner)];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25088))] = compute_local[((ff_inner_inner_inner + 8))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 1))] = compute_local[((ff_inner_inner_inner + 2))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25089))] = compute_local[((ff_inner_inner_inner + 10))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 2))] = compute_local[((ff_inner_inner_inner + 4))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25090))] = compute_local[((ff_inner_inner_inner + 12))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 3))] = compute_local[((ff_inner_inner_inner + 6))];
    compute[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 3136)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + 25091))] = compute_local[((ff_inner_inner_inner + 14))];
  }
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(14,1,4);
    dim3 block(1,56,4);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


