#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 32
#define N 32
#define H 28
#define W 28

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[72];
  float pad_temp_shared_local[24];
  float kernel_shared_local[18];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 28) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 28) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + (((((int)threadIdx.x) * 28) / 96) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 28) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 28) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 1) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 1) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 1) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 1) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 2) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 2) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 2) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 2) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 3) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 3) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 3) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 3) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 4) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 4) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 4) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 4) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 5) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 5) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 5) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 5) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 6) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 6) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 6) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 6) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 7) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 7) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 7) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 7) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 7) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 8) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 8) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 8) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 8) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 8) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 9))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 9) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 9) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 9) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 9) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 9) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 10))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 10) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 10) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 10) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 10) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 10) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 11))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 11) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 11) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 11) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 11) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 11) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 12) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 12) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 12) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 12) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 12) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 12) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 13) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 13) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 13) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 13) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 13) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 13) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 14) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 14) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 14) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 14) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 14) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 14) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 15) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 15) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 15) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 15) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 15) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 15) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 16) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 16) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 28) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 16) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 16) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 28) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 17) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 17) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 17) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 17) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 18))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 18) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 18) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 18) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 18) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 19))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 19) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 19) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 19) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 19) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 20))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 20) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 20) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 20) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 20) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 21))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 21) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 21) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 5) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 21) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 21) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 22))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 22) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 22) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 6) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 22) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 22) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 23))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 23) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 23) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 7) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 23) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 23) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 7) & 15)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 24) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 24) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 744) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 24))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 24) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 24) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 8) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 24) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 24) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 8) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 25) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 25) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 743) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 25))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 25) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 25) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 9) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 25) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 25) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 9) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 26) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 26) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 742) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 26))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 26) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 26) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 10) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 26) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 26) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 10) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 28) + 27) / 96)) < 8) {
      if (((((int)threadIdx.y) * 12) + (((((int)threadIdx.x) * 28) + 27) >> 4)) < 48) {
        if (((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) < 741) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.y) * 192) + (((int)threadIdx.x) * 28)) + 27))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 27) % 96) >> 4))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 28) + 27) % 96) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 28) + 11) & 15)) < 29)) ? data[(((((((((rc_outer * 6272) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 28) + 27) / 96) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 28) + 27) % 96) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 28) + 11) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 72) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[(((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)))] = kernel[(((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 71) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) + 1))] = kernel[((((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 8) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 24) {
        if (((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) < 70) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.y) * 18) + (((int)threadIdx.x) * 3)) + 2))] = kernel[((((((((int)blockIdx.z) * 288) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 3)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 3))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 16))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 17))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 18))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 19))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 33))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 34))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 35))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 99))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 112))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 113))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 114))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 115))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 129))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 130))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 131))];
    kernel_shared_local[(0)] = kernel_shared[(0)];
    kernel_shared_local[(1)] = kernel_shared[(1)];
    kernel_shared_local[(2)] = kernel_shared[(2)];
    kernel_shared_local[(3)] = kernel_shared[(3)];
    kernel_shared_local[(4)] = kernel_shared[(4)];
    kernel_shared_local[(5)] = kernel_shared[(5)];
    kernel_shared_local[(6)] = kernel_shared[(6)];
    kernel_shared_local[(7)] = kernel_shared[(7)];
    kernel_shared_local[(8)] = kernel_shared[(8)];
    kernel_shared_local[(9)] = kernel_shared[(9)];
    kernel_shared_local[(10)] = kernel_shared[(10)];
    kernel_shared_local[(11)] = kernel_shared[(11)];
    kernel_shared_local[(12)] = kernel_shared[(12)];
    kernel_shared_local[(13)] = kernel_shared[(13)];
    kernel_shared_local[(14)] = kernel_shared[(14)];
    kernel_shared_local[(15)] = kernel_shared[(15)];
    kernel_shared_local[(16)] = kernel_shared[(16)];
    kernel_shared_local[(17)] = kernel_shared[(17)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 195))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 208))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 209))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 210))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 211))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 225))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 226))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 227))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 291))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 304))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 305))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 306))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 307))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 321))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 322))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 323))];
    kernel_shared_local[(0)] = kernel_shared[(18)];
    kernel_shared_local[(1)] = kernel_shared[(19)];
    kernel_shared_local[(2)] = kernel_shared[(20)];
    kernel_shared_local[(3)] = kernel_shared[(21)];
    kernel_shared_local[(4)] = kernel_shared[(22)];
    kernel_shared_local[(5)] = kernel_shared[(23)];
    kernel_shared_local[(6)] = kernel_shared[(24)];
    kernel_shared_local[(7)] = kernel_shared[(25)];
    kernel_shared_local[(8)] = kernel_shared[(26)];
    kernel_shared_local[(9)] = kernel_shared[(27)];
    kernel_shared_local[(10)] = kernel_shared[(28)];
    kernel_shared_local[(11)] = kernel_shared[(29)];
    kernel_shared_local[(12)] = kernel_shared[(30)];
    kernel_shared_local[(13)] = kernel_shared[(31)];
    kernel_shared_local[(14)] = kernel_shared[(32)];
    kernel_shared_local[(15)] = kernel_shared[(33)];
    kernel_shared_local[(16)] = kernel_shared[(34)];
    kernel_shared_local[(17)] = kernel_shared[(35)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 386))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 387))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 400))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 401))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 402))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 403))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 416))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 417))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 418))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 419))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 483))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 496))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 497))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 498))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 499))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 512))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 513))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 514))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 515))];
    kernel_shared_local[(0)] = kernel_shared[(36)];
    kernel_shared_local[(1)] = kernel_shared[(37)];
    kernel_shared_local[(2)] = kernel_shared[(38)];
    kernel_shared_local[(3)] = kernel_shared[(39)];
    kernel_shared_local[(4)] = kernel_shared[(40)];
    kernel_shared_local[(5)] = kernel_shared[(41)];
    kernel_shared_local[(6)] = kernel_shared[(42)];
    kernel_shared_local[(7)] = kernel_shared[(43)];
    kernel_shared_local[(8)] = kernel_shared[(44)];
    kernel_shared_local[(9)] = kernel_shared[(45)];
    kernel_shared_local[(10)] = kernel_shared[(46)];
    kernel_shared_local[(11)] = kernel_shared[(47)];
    kernel_shared_local[(12)] = kernel_shared[(48)];
    kernel_shared_local[(13)] = kernel_shared[(49)];
    kernel_shared_local[(14)] = kernel_shared[(50)];
    kernel_shared_local[(15)] = kernel_shared[(51)];
    kernel_shared_local[(16)] = kernel_shared[(52)];
    kernel_shared_local[(17)] = kernel_shared[(53)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 576))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 577))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 578))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 579))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 592))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 593))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 594))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 595))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 608))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 609))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 610))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 611))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 672))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 673))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 674))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 675))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 688))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 689))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 690))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 691))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 704))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 705))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 706))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(54)];
    kernel_shared_local[(1)] = kernel_shared[(55)];
    kernel_shared_local[(2)] = kernel_shared[(56)];
    kernel_shared_local[(3)] = kernel_shared[(57)];
    kernel_shared_local[(4)] = kernel_shared[(58)];
    kernel_shared_local[(5)] = kernel_shared[(59)];
    kernel_shared_local[(6)] = kernel_shared[(60)];
    kernel_shared_local[(7)] = kernel_shared[(61)];
    kernel_shared_local[(8)] = kernel_shared[(62)];
    kernel_shared_local[(9)] = kernel_shared[(63)];
    kernel_shared_local[(10)] = kernel_shared[(64)];
    kernel_shared_local[(11)] = kernel_shared[(65)];
    kernel_shared_local[(12)] = kernel_shared[(66)];
    kernel_shared_local[(13)] = kernel_shared[(67)];
    kernel_shared_local[(14)] = kernel_shared[(68)];
    kernel_shared_local[(15)] = kernel_shared[(69)];
    kernel_shared_local[(16)] = kernel_shared[(70)];
    kernel_shared_local[(17)] = kernel_shared[(71)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(17)]));
  }
  compute[((((((((int)blockIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
}




float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(2,7,32);
    dim3 block(7,4,1);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


