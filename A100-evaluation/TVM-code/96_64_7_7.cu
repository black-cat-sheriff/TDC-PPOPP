#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 96
#define N 64
#define H 7
#define W 7

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[504];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[24];
  float kernel_shared_local[24];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = ((((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 9) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 1) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 2) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 3) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 4) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 5) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 6) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 9) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 9) + 1) % 7))) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 8))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 15))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 18))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 21))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 24))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 27))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 30))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = (((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = (((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = (((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = (((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = (((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = (((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = (((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = (((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 7))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = (((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 7))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 16))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 19))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 22))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 25))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 28))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 31))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)))] = ((((1 <= ((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 9) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (((((int)threadIdx.x) * 9) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + (((((int)threadIdx.x) * 9) / 21) * 49)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 9) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 1))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 1) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 1) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 1) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 1) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 2))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 2) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 2) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 2) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 2) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 3))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 3) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 3) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 3) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 3) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 4))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 4) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 4) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 4) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 4) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 5))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 5) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 5) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 5) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 5) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 6))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 6) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 6) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 6) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 6) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 7))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 7) % 21) / 7) + ((int)blockIdx.y)) < 8)) && (((((int)threadIdx.x) * 9) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 7) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 7) % 21)) - 6))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 63) + (((int)threadIdx.x) * 9)) + 8))] = ((((1 <= (((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 9) + 8) % 21) / 7) + ((int)blockIdx.y)) < 8)) && ((((((int)threadIdx.x) * 9) + 1) % 7) < 6)) ? data[(((((((rc_outer * 1176) + (((int)threadIdx.z) * 147)) + ((((((int)threadIdx.x) * 9) + 8) / 21) * 49)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 9) + 8) % 21)) - 6))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 17))];
    if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 20))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 23))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 26))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 29))];
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 864)) + (rc_outer * 216)) + (((int)threadIdx.x) * 33)) + 32))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 231))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 273))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 294))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 336))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 357))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 399))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 420))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 462))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 483))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 238))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 259))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 301))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 322))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 364))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 385))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 427))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 448))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 490))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 224))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 266))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 287))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 329))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.x) + 350))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.x) + 392))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.x) + 413))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.x) + 455))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.x) + 476))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(1,7,8);
    dim3 block(7,1,8);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


