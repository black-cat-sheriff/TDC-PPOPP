#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 64
#define N 32
#define H 224
#define W 224

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[112];
  __shared__ float pad_temp_shared[7424];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[42];
  float kernel_shared_local[24];
  #pragma unroll
  for (int ff_c_init = 0; ff_c_init < 8; ++ff_c_init) {
    #pragma unroll
    for (int yy_c_init = 0; yy_c_init < 14; ++yy_c_init) {
      compute_local[(((ff_c_init * 14) + yy_c_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 34; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 928)) < 8) {
        if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 58)) < 128) {
          if ((((((int)threadIdx.z) * 1856) + (((int)threadIdx.x) * 34)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 7424) {
            if (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 1856) {
              pad_temp_shared[((((((int)threadIdx.z) * 1856) + (((int)threadIdx.x) * 34)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 14) + ((((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 928) / 58))) && (((((int)blockIdx.y) * 14) + ((((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 928) / 58)) < 225)) && (1 <= ((((int)blockIdx.x) * 56) + (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 58)))) && (((((int)blockIdx.x) * 56) + (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 58)) < 225)) ? data[(((((((((rc_outer * 401408) + (((int)threadIdx.z) * 100352)) + ((((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 928) * 50176)) + (((int)blockIdx.y) * 3136)) + (((((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 928) / 58) * 224)) + (((int)blockIdx.x) * 56)) + (((((int)threadIdx.x) * 34) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 58)) - 225))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 72)) < 32) {
        if (((((int)threadIdx.z) * 64) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 9)) < 256) {
          if (((((int)threadIdx.z) * 192) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 3)) < 768) {
            if ((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 11)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 2304) {
              if (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 576) {
                kernel_shared[((((((int)threadIdx.z) * 576) + (((int)threadIdx.x) * 11)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((int)threadIdx.z) * 4608) + ((((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 72) * 576)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 72)))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
      for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
        #pragma unroll
        for (int ax2 = 0; ax2 < 14; ++ax2) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 3; ++ax3) {
            pad_temp_shared_local[(((ax2 * 3) + ax3))] = pad_temp_shared[((((((rc_inner_outer * 928) + (ax2 * 58)) + (ry_inner_outer * 58)) + ax3) + ((int)threadIdx.x)))];
          }
        }
        #pragma unroll
        for (int ax0 = 0; ax0 < 8; ++ax0) {
          #pragma unroll
          for (int ax31 = 0; ax31 < 3; ++ax31) {
            kernel_shared_local[(((ax0 * 3) + ax31))] = kernel_shared[((((((((int)threadIdx.z) * 576) + (ax0 * 72)) + (rc_inner_outer * 9)) + (ry_inner_outer * 3)) + ax31))];
          }
        }
        #pragma unroll
        for (int rx_inner_inner = 0; rx_inner_inner < 3; ++rx_inner_inner) {
          #pragma unroll
          for (int ff_c = 0; ff_c < 8; ++ff_c) {
            #pragma unroll
            for (int yy_c = 0; yy_c < 14; ++yy_c) {
              compute_local[(((ff_c * 14) + yy_c))] = (compute_local[(((ff_c * 14) + yy_c))] + (pad_temp_shared_local[(((yy_c * 3) + rx_inner_inner))] * kernel_shared_local[(((ff_c * 3) + rx_inner_inner))]));
            }
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 8; ++ff_inner_inner_inner) {
    #pragma unroll
    for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 14; ++yy_inner_inner_inner) {
      compute[(((((((((int)threadIdx.z) * 401408) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 3136)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 56)) + ((int)threadIdx.x)))] = compute_local[(((ff_inner_inner_inner * 14) + yy_inner_inner_inner))];
    }
  }
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(4,16,1);
    dim3 block(56,1,4);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


