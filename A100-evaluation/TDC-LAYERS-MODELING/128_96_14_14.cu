#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 2
#define TC 16
#define C 128
#define N 96
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[32];
  __shared__ float kernel_shared[128];
  float pad_temp_shared_local[8];
  float kernel_shared_local[4];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = (((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) && (1 <= ((int)blockIdx.x))) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 15))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 14))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = (((((((int)blockIdx.y) * 2) + ry_outer) < 14) && (1 <= ((int)blockIdx.x))) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 1))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[(((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 9))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 18))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 27))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 36))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 45))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 54))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 63))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1152))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1161))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1170))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1179))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1188))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1197))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1206))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1215))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 14))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 13))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 1))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 10))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 19))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 28))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 37))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 46))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 55))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 64))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1153))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1162))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1171))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1180))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1189))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1198))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1207))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1216))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 13))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = (((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) && (((int)blockIdx.x) < 6)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 12))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 1))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = (((((((int)blockIdx.y) * 2) + ry_outer) < 14) && (((int)blockIdx.x) < 6)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 2))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 11))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 20))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 29))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 38))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 47))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 56))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 65))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1154))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1163))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1172))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1181))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1190))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1199))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1208))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1217))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1568))] = compute_local[(4)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1569))] = compute_local[(5)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 14))] = compute_local[(2)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1582))] = compute_local[(6)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 15))] = compute_local[(3)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1583))] = compute_local[(7)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(0-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(1-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(2-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(3-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(0-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(1-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(2-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(3-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(0-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(1-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(2-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(3-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(7,7,6);

        dim3 block(1,1,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    hipMemset(device_out, 0, sizeof(float)*N*H*W);
    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


