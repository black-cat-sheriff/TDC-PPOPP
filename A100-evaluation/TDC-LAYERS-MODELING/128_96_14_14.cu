#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 2
#define TC 16
#define C 128
#define N 96
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[32];
  __shared__ float kernel_shared[128];
  float pad_temp_shared_local[8];
  float kernel_shared_local[4];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = (((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) && (1 <= ((int)blockIdx.x))) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 15))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 14))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = (((((((int)blockIdx.y) * 2) + ry_outer) < 14) && (1 <= ((int)blockIdx.x))) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 1))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[(((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 9))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 18))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 27))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 36))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 45))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 54))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 63))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1152))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1161))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1170))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1179))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1188))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1197))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1206))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1215))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 14))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 13))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 1))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 10))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 19))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 28))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 37))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 46))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 55))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 64))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1153))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1162))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1171))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1180))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1189))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1198))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1207))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1216))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 4))] = ((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 13))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 1))] = (((1 <= ((((int)blockIdx.y) * 2) + ry_outer)) && (((int)blockIdx.x) < 6)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) - 12))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 2))] = ((((((int)blockIdx.y) * 2) + ry_outer) < 14) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 1))] : 0.000000e+00f);
      pad_temp_shared[(((((int)threadIdx.z) * 4) + 3))] = (((((((int)blockIdx.y) * 2) + ry_outer) < 14) && (((int)blockIdx.x) < 6)) ? data[(((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + (((int)blockIdx.x) * 2)) + 2))] : 0.000000e+00f);
      kernel_shared[((((int)threadIdx.z) * 16))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 1))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 11))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 2))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 20))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 3))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 29))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 4))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 38))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 5))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 47))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 6))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 56))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 7))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 65))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 8))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1154))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 9))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1163))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 10))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1172))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 11))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1181))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 12))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1190))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 13))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1199))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 14))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1208))];
      kernel_shared[(((((int)threadIdx.z) * 16) + 15))] = kernel[((((((((int)blockIdx.z) * 18432) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (ry_outer * 3)) + 1217))];
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(0)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(1)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(2)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(3)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(4)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(5)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(6)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(7)];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 8))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 64))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 1))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(8)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(9)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(10)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(11)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(12)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(13)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(14)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(15)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 2))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 66))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 3))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 67))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(16)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(17)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(18)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(19)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(20)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(21)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(22)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(23)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 4))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 68))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 5))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 69))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(24)];
      pad_temp_shared_local[(1)] = pad_temp_shared[(25)];
      pad_temp_shared_local[(2)] = pad_temp_shared[(26)];
      pad_temp_shared_local[(3)] = pad_temp_shared[(27)];
      pad_temp_shared_local[(4)] = pad_temp_shared[(28)];
      pad_temp_shared_local[(5)] = pad_temp_shared[(29)];
      pad_temp_shared_local[(6)] = pad_temp_shared[(30)];
      pad_temp_shared_local[(7)] = pad_temp_shared[(31)];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 8) + 6))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 8) + 70))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 8) + 7))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 8) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1568))] = compute_local[(4)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1569))] = compute_local[(5)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 14))] = compute_local[(2)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1582))] = compute_local[(6)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 15))] = compute_local[(3)];
  compute[((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 2)) + 1583))] = compute_local[(7)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(7,7,6);

                dim3 block(1,1,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


