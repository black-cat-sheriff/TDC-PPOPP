#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 4
#define TC 16
#define C 160
#define N 96
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[1920];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[64];
  float kernel_shared_local[48];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 10; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 35) % 120) / 30))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 35) % 120) / 30)) < 29)) && (1 <= ((((int)threadIdx.x) * 35) % 30))) && (((((int)threadIdx.x) * 35) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + (((((int)threadIdx.x) * 35) / 120) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.x) * 35) % 120) / 30) * 28)) + ((((int)threadIdx.x) * 35) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 1) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 1) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 1) % 30))) && ((((((int)threadIdx.x) * 35) + 1) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 1) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 1) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 1) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 2) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 2) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 2) % 30))) && ((((((int)threadIdx.x) * 35) + 2) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 2) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 2) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 2) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 3) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 3) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 3) % 30))) && ((((((int)threadIdx.x) * 35) + 3) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 3) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 3) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 3) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 4) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 4) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 4) % 30))) && ((((((int)threadIdx.x) * 35) + 4) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 4) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 4) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 4) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 5) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 5) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 5) % 30))) && ((((((int)threadIdx.x) * 35) + 5) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 5) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 5) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 5) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 6) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 6) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 6) % 30))) && ((((((int)threadIdx.x) * 35) + 6) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 6) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 6) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 6) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 7) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 7) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 7) % 30))) && ((((((int)threadIdx.x) * 35) + 7) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 7) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 7) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 7) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 8) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 8) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 8) % 30))) && ((((((int)threadIdx.x) * 35) + 8) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 8) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 8) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 8) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 9))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 9) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 9) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 9) % 30))) && ((((((int)threadIdx.x) * 35) + 9) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 9) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 9) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 9) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 10))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 10) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 10) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 10) % 30))) && ((((((int)threadIdx.x) * 35) + 10) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 10) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 10) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 10) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 11))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 11) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 11) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 11) % 30))) && ((((((int)threadIdx.x) * 35) + 11) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 11) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 11) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 11) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 12) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 12) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 12) % 30))) && ((((((int)threadIdx.x) * 35) + 12) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 12) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 12) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 12) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 13) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 13) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 13) % 30))) && ((((((int)threadIdx.x) * 35) + 13) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 13) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 13) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 13) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 14) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 14) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 14) % 30))) && ((((((int)threadIdx.x) * 35) + 14) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 14) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 14) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 14) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 15) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 15) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 15) % 30))) && ((((((int)threadIdx.x) * 35) + 15) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 15) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 15) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 15) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 16) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 16) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 16) % 30))) && ((((((int)threadIdx.x) * 35) + 16) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 16) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 16) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 16) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 17) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 17) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 17) % 30))) && ((((((int)threadIdx.x) * 35) + 17) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 17) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 17) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 17) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 18))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 18) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 18) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 18) % 30))) && ((((((int)threadIdx.x) * 35) + 18) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 18) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 18) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 18) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 19))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 19) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 19) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 19) % 30))) && ((((((int)threadIdx.x) * 35) + 19) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 19) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 19) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 19) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 20))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 20) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 20) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 20) % 30))) && ((((((int)threadIdx.x) * 35) + 20) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 20) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 20) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 20) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 21))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 21) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 21) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 21) % 30))) && ((((((int)threadIdx.x) * 35) + 21) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 21) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 21) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 21) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 22))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 22) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 22) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 22) % 30))) && ((((((int)threadIdx.x) * 35) + 22) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 22) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 22) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 22) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 23))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 23) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 23) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 23) % 30))) && ((((((int)threadIdx.x) * 35) + 23) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 23) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 23) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 23) % 30)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 24))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 24) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 24) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 24) % 30))) && ((((((int)threadIdx.x) * 35) + 24) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 24) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 24) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 24) % 30)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 25) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 25) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1895) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 25))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 25) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 25) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 25) % 30))) && ((((((int)threadIdx.x) * 35) + 25) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 25) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 25) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 25) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 26) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 26) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1894) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 26))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 26) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 26) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 26) % 30))) && ((((((int)threadIdx.x) * 35) + 26) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 26) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 26) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 26) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 27) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 27) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1893) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 27))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 27) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 27) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 27) % 30))) && ((((((int)threadIdx.x) * 35) + 27) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 27) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 27) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 27) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 28) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 28) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1892) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 28))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 28) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 28) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 28) % 30))) && ((((((int)threadIdx.x) * 35) + 28) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 28) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 28) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 28) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 29) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 29) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1891) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 29))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 29) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 29) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 29) % 30))) && ((((((int)threadIdx.x) * 35) + 29) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 29) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 29) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 29) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 30) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 35) / 30)) < 63) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1890) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 30))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 30) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 30) % 120) / 30)) < 29)) && (1 <= ((((int)threadIdx.x) * 35) % 30))) && (((((int)threadIdx.x) * 35) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 30) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 30) % 120) / 30) * 28)) + ((((int)threadIdx.x) * 35) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 31) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 31) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1889) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 31))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 31) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 31) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 1) % 30))) && ((((((int)threadIdx.x) * 35) + 1) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 31) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 31) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 1) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 32) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 32) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1888) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 32))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 32) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 32) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 2) % 30))) && ((((((int)threadIdx.x) * 35) + 2) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 32) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 32) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 2) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 33) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 33) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1887) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 33))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 33) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 33) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 3) % 30))) && ((((((int)threadIdx.x) * 35) + 3) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 33) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 33) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 3) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 35) + 34) / 120)) < 16) {
      if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 35) + 34) / 30)) < 64) {
        if (((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) < 1886) {
          if (((int)threadIdx.x) < 13) {
            pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.x) * 35)) + 34))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 34) % 120) / 30))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 35) + 34) % 120) / 30)) < 29)) && (1 <= (((((int)threadIdx.x) * 35) + 4) % 30))) && ((((((int)threadIdx.x) * 35) + 4) % 30) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 3136)) + ((((((int)threadIdx.x) * 35) + 34) / 120) * 784)) + (((int)blockIdx.y) * 56)) + (((((((int)threadIdx.x) * 35) + 34) % 120) / 30) * 28)) + (((((int)threadIdx.x) * 35) + 4) % 30)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + (((((int)threadIdx.x) * 7) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 1))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + (((((int)threadIdx.x) * 7) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 2))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + (((((int)threadIdx.x) * 7) / 48) * 1440)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 4))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 5))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 7))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 8))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 10))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 11))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 13))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 14))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 379) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1137) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 379) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1136) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 16))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 379) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1135) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 17))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1134) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1133) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 19))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 8) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 1132) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 20))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 2880)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1440)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 1))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 2))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 3))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 30))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 31))];
        pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 32))];
        pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 33))];
        pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 120))];
        pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 121))];
        pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 122))];
        pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 123))];
        pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 150))];
        pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 151))];
        pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 152))];
        pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 153))];
        pad_temp_shared_local[(16)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 240))];
        pad_temp_shared_local[(17)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 241))];
        pad_temp_shared_local[(18)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 242))];
        pad_temp_shared_local[(19)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 243))];
        pad_temp_shared_local[(20)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 270))];
        pad_temp_shared_local[(21)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 271))];
        pad_temp_shared_local[(22)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 272))];
        pad_temp_shared_local[(23)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 273))];
        pad_temp_shared_local[(24)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 360))];
        pad_temp_shared_local[(25)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 361))];
        pad_temp_shared_local[(26)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 362))];
        pad_temp_shared_local[(27)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 363))];
        pad_temp_shared_local[(28)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 390))];
        pad_temp_shared_local[(29)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 391))];
        pad_temp_shared_local[(30)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 392))];
        pad_temp_shared_local[(31)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 393))];
        pad_temp_shared_local[(32)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 480))];
        pad_temp_shared_local[(33)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 481))];
        pad_temp_shared_local[(34)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 482))];
        pad_temp_shared_local[(35)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 483))];
        pad_temp_shared_local[(36)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 510))];
        pad_temp_shared_local[(37)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 511))];
        pad_temp_shared_local[(38)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 512))];
        pad_temp_shared_local[(39)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 513))];
        pad_temp_shared_local[(40)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 600))];
        pad_temp_shared_local[(41)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 601))];
        pad_temp_shared_local[(42)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 602))];
        pad_temp_shared_local[(43)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 603))];
        pad_temp_shared_local[(44)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 630))];
        pad_temp_shared_local[(45)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 631))];
        pad_temp_shared_local[(46)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 632))];
        pad_temp_shared_local[(47)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 633))];
        pad_temp_shared_local[(48)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 720))];
        pad_temp_shared_local[(49)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 721))];
        pad_temp_shared_local[(50)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 722))];
        pad_temp_shared_local[(51)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 723))];
        pad_temp_shared_local[(52)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 750))];
        pad_temp_shared_local[(53)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 751))];
        pad_temp_shared_local[(54)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 752))];
        pad_temp_shared_local[(55)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 753))];
        pad_temp_shared_local[(56)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 840))];
        pad_temp_shared_local[(57)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 841))];
        pad_temp_shared_local[(58)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 842))];
        pad_temp_shared_local[(59)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 843))];
        pad_temp_shared_local[(60)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 870))];
        pad_temp_shared_local[(61)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 871))];
        pad_temp_shared_local[(62)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 872))];
        pad_temp_shared_local[(63)] = pad_temp_shared[(((((rc_inner_outer * 960) + (ry_inner_outer * 30)) + (((int)threadIdx.x) * 2)) + 873))];
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)))];
        kernel_shared_local[(24)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 576))];
        kernel_shared_local[(1)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 1))];
        kernel_shared_local[(25)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 577))];
        kernel_shared_local[(2)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 2))];
        kernel_shared_local[(26)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 578))];
        kernel_shared_local[(3)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 9))];
        kernel_shared_local[(27)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 585))];
        kernel_shared_local[(4)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 10))];
        kernel_shared_local[(28)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 586))];
        kernel_shared_local[(5)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 11))];
        kernel_shared_local[(29)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 587))];
        kernel_shared_local[(6)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 18))];
        kernel_shared_local[(30)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 594))];
        kernel_shared_local[(7)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 19))];
        kernel_shared_local[(31)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 595))];
        kernel_shared_local[(8)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 20))];
        kernel_shared_local[(32)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 596))];
        kernel_shared_local[(9)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 27))];
        kernel_shared_local[(33)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 603))];
        kernel_shared_local[(10)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 28))];
        kernel_shared_local[(34)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 604))];
        kernel_shared_local[(11)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 29))];
        kernel_shared_local[(35)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 605))];
        kernel_shared_local[(12)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 36))];
        kernel_shared_local[(36)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 612))];
        kernel_shared_local[(13)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 37))];
        kernel_shared_local[(37)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 613))];
        kernel_shared_local[(14)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 38))];
        kernel_shared_local[(38)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 614))];
        kernel_shared_local[(15)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 45))];
        kernel_shared_local[(39)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 621))];
        kernel_shared_local[(16)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 46))];
        kernel_shared_local[(40)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 622))];
        kernel_shared_local[(17)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 47))];
        kernel_shared_local[(41)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 623))];
        kernel_shared_local[(18)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 54))];
        kernel_shared_local[(42)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 630))];
        kernel_shared_local[(19)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 55))];
        kernel_shared_local[(43)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 631))];
        kernel_shared_local[(20)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 56))];
        kernel_shared_local[(44)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 632))];
        kernel_shared_local[(21)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 63))];
        kernel_shared_local[(45)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 639))];
        kernel_shared_local[(22)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 64))];
        kernel_shared_local[(46)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 640))];
        kernel_shared_local[(23)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 65))];
        kernel_shared_local[(47)] = kernel_shared[(((((((int)threadIdx.z) * 144) + (rc_inner_outer * 72)) + (ry_inner_outer * 3)) + 641))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(24)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(24)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(24)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(24)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(25)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(25)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(25)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(25)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(26)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(26)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(2)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(26)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(2)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(26)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(27)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(27)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(27)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(27)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(4)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(28)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(28)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(4)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(28)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(4)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(28)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(29)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(29)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(5)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(29)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(5)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(29)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(6)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(30)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(6)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(30)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(6)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(30)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(6)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(30)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(7)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(31)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(7)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(31)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(7)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(31)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(7)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(31)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(8)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(32)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(8)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(32)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(8)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(32)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(8)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(32)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(9)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(33)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(9)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(33)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(9)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(33)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(9)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(33)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(10)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(34)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(10)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(34)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(10)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(34)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(10)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(34)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(11)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(35)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(11)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(35)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(11)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(35)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(11)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(35)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(12)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(36)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(12)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(36)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(12)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(36)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(12)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(36)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(13)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(37)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(13)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(37)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(13)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(37)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(13)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(37)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(14)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(38)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(14)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(38)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(14)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(38)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(14)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(38)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(15)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(39)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(15)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(39)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(15)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(39)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(15)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(39)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(16)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(40)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(16)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(40)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(16)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(40)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(16)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(40)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(17)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(41)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(17)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(41)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(17)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(41)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(17)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(41)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(18)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(42)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(18)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(42)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(18)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(42)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(18)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(42)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(19)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(43)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(19)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(43)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(19)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(43)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(19)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(43)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(20)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(44)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(20)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(44)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(20)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(44)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(20)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(44)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(21)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(45)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(21)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(45)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(21)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(45)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(21)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(45)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(22)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(46)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(22)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(46)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(22)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(46)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(22)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(46)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(23)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(47)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(23)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(47)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(23)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(47)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(23)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(47)]));
      }
    }
  }
  compute[(((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 3136))] = compute_local[(4)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 3137))] = compute_local[(5)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 28))] = compute_local[(2)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 3164))] = compute_local[(6)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 29))] = compute_local[(3)];
  compute[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.x) * 2)) + 3165))] = compute_local[(7)];
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,14,12);

                dim3 block(14,1,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


