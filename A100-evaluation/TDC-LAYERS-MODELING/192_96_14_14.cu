#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 2
#define TC 16
#define C 192
#define N 96
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[6144];
  __shared__ float kernel_shared[5184];
  float pad_temp_shared_local[2];
  float kernel_shared_local[4];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 74) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 74) & 63) >> 4)) < 15)) && (1 <= ((((int)threadIdx.x) * 74) & 15))) && (((((int)threadIdx.x) * 74) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + (((((int)threadIdx.x) * 74) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 74) & 63) >> 4) * 14)) + ((((int)threadIdx.x) * 74) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 1) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 1) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 1) & 15))) && ((((((int)threadIdx.x) * 74) + 1) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 1) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 1) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 2) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 2) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 2) & 15))) && ((((((int)threadIdx.x) * 74) + 2) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 2) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 2) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 3) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 3) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 3) & 15))) && ((((((int)threadIdx.x) * 74) + 3) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 3) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 3) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 4) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 4) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 4) & 15))) && ((((((int)threadIdx.x) * 74) + 4) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 4) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 4) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 5) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 5) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 5) & 15))) && ((((((int)threadIdx.x) * 74) + 5) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 5) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 5) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 6) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 6) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 6) & 15))) && ((((((int)threadIdx.x) * 74) + 6) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 6) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 6) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 7) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 7) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 7) & 15))) && ((((((int)threadIdx.x) * 74) + 7) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 7) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 7) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 8) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 8) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 8) & 15))) && ((((((int)threadIdx.x) * 74) + 8) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 8) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 8) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 8) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 9))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 9) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 9) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 9) & 15))) && ((((((int)threadIdx.x) * 74) + 9) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 9) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 9) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 9) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 10))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 10) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 10) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 10) & 15))) && ((((((int)threadIdx.x) * 74) + 10) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 10) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 10) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 10) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 11))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 11) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 11) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 11) & 15))) && ((((((int)threadIdx.x) * 74) + 11) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 11) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 11) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 11) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 12) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 12) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 12) & 15))) && ((((((int)threadIdx.x) * 74) + 12) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 12) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 12) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 12) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 13) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 13) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 13) & 15))) && ((((((int)threadIdx.x) * 74) + 13) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 13) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 13) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 13) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 14) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 14) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 14) & 15))) && ((((((int)threadIdx.x) * 74) + 14) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 14) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 14) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 14) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 15) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 15) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 15) & 15))) && ((((((int)threadIdx.x) * 74) + 15) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 15) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 15) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 15) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 16) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 16) & 63) >> 4)) < 15)) && (1 <= ((((int)threadIdx.x) * 74) & 15))) && (((((int)threadIdx.x) * 74) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 16) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 16) & 63) >> 4) * 14)) + ((((int)threadIdx.x) * 74) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 17) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 17) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 1) & 15))) && ((((((int)threadIdx.x) * 74) + 1) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 17) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 17) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 18))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 18) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 18) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 2) & 15))) && ((((((int)threadIdx.x) * 74) + 2) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 18) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 18) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 19))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 19) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 19) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 3) & 15))) && ((((((int)threadIdx.x) * 74) + 3) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 19) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 19) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 20))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 20) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 20) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 4) & 15))) && ((((((int)threadIdx.x) * 74) + 4) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 20) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 20) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 21))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 21) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 21) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 5) & 15))) && ((((((int)threadIdx.x) * 74) + 5) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 21) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 21) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 22))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 22) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 22) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 6) & 15))) && ((((((int)threadIdx.x) * 74) + 6) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 22) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 22) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 23))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 23) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 23) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 7) & 15))) && ((((((int)threadIdx.x) * 74) + 7) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 23) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 23) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 24))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 24) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 24) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 8) & 15))) && ((((((int)threadIdx.x) * 74) + 8) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 24) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 24) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 8) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 25))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 25) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 25) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 9) & 15))) && ((((((int)threadIdx.x) * 74) + 9) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 25) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 25) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 9) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 26))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 26) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 26) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 10) & 15))) && ((((((int)threadIdx.x) * 74) + 10) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 26) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 26) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 10) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 27))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 27) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 27) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 11) & 15))) && ((((((int)threadIdx.x) * 74) + 11) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 27) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 27) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 11) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 28))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 28) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 28) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 12) & 15))) && ((((((int)threadIdx.x) * 74) + 12) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 28) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 28) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 12) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 29))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 29) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 29) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 13) & 15))) && ((((((int)threadIdx.x) * 74) + 13) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 29) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 29) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 13) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 30))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 30) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 30) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 14) & 15))) && ((((((int)threadIdx.x) * 74) + 14) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 30) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 30) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 14) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 31))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 31) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 31) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 15) & 15))) && ((((((int)threadIdx.x) * 74) + 15) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 31) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 31) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 15) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 32))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 32) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 32) & 63) >> 4)) < 15)) && (1 <= ((((int)threadIdx.x) * 74) & 15))) && (((((int)threadIdx.x) * 74) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 32) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 32) & 63) >> 4) * 14)) + ((((int)threadIdx.x) * 74) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 33))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 33) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 33) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 1) & 15))) && ((((((int)threadIdx.x) * 74) + 1) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 33) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 33) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 34))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 34) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 34) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 2) & 15))) && ((((((int)threadIdx.x) * 74) + 2) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 34) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 34) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 35))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 35) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 35) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 3) & 15))) && ((((((int)threadIdx.x) * 74) + 3) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 35) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 35) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 36))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 36) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 36) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 4) & 15))) && ((((((int)threadIdx.x) * 74) + 4) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 36) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 36) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 37))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 37) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 37) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 5) & 15))) && ((((((int)threadIdx.x) * 74) + 5) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 37) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 37) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 38))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 38) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 38) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 6) & 15))) && ((((((int)threadIdx.x) * 74) + 6) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 38) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 38) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 39))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 39) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 39) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 7) & 15))) && ((((((int)threadIdx.x) * 74) + 7) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 39) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 39) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 40))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 40) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 40) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 8) & 15))) && ((((((int)threadIdx.x) * 74) + 8) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 40) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 40) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 8) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 41))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 41) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 41) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 9) & 15))) && ((((((int)threadIdx.x) * 74) + 9) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 41) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 41) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 9) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 42))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 42) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 42) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 10) & 15))) && ((((((int)threadIdx.x) * 74) + 10) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 42) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 42) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 10) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 43))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 43) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 43) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 11) & 15))) && ((((((int)threadIdx.x) * 74) + 11) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 43) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 43) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 11) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 44))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 44) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 44) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 12) & 15))) && ((((((int)threadIdx.x) * 74) + 12) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 44) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 44) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 12) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 45))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 45) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 45) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 13) & 15))) && ((((((int)threadIdx.x) * 74) + 13) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 45) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 45) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 13) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 46))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 46) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 46) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 14) & 15))) && ((((((int)threadIdx.x) * 74) + 14) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 46) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 46) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 14) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 47))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 47) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 47) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 15) & 15))) && ((((((int)threadIdx.x) * 74) + 15) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 47) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 47) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 15) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 48))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 48) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 48) & 63) >> 4)) < 15)) && (1 <= ((((int)threadIdx.x) * 74) & 15))) && (((((int)threadIdx.x) * 74) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 48) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 48) & 63) >> 4) * 14)) + ((((int)threadIdx.x) * 74) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 49))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 49) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 49) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 1) & 15))) && ((((((int)threadIdx.x) * 74) + 1) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 49) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 49) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 50))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 50) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 50) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 2) & 15))) && ((((((int)threadIdx.x) * 74) + 2) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 50) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 50) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 51))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 51) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 51) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 3) & 15))) && ((((((int)threadIdx.x) * 74) + 3) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 51) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 51) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 52))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 52) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 52) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 4) & 15))) && ((((((int)threadIdx.x) * 74) + 4) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 52) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 52) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 53))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 53) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 53) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 5) & 15))) && ((((((int)threadIdx.x) * 74) + 5) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 53) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 53) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 54))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 54) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 54) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 6) & 15))) && ((((((int)threadIdx.x) * 74) + 6) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 54) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 54) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 55))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 55) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 55) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 7) & 15))) && ((((((int)threadIdx.x) * 74) + 7) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 55) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 55) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 56))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 56) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 56) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 8) & 15))) && ((((((int)threadIdx.x) * 74) + 8) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 56) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 56) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 8) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 57))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 57) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 57) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 9) & 15))) && ((((((int)threadIdx.x) * 74) + 9) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 57) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 57) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 9) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 58))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 58) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 58) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 10) & 15))) && ((((((int)threadIdx.x) * 74) + 10) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 58) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 58) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 10) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 59))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 59) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 59) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 11) & 15))) && ((((((int)threadIdx.x) * 74) + 11) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 59) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 59) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 11) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 60))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 60) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 60) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 12) & 15))) && ((((((int)threadIdx.x) * 74) + 12) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 60) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 60) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 12) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 61))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 61) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 61) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 13) & 15))) && ((((((int)threadIdx.x) * 74) + 13) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 61) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 61) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 13) & 15)) - 15))] : 0.000000e+00f);
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 62) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 62) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6082) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1986) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 62))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 62) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 62) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 14) & 15))) && ((((((int)threadIdx.x) * 74) + 14) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 62) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 62) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 14) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 63) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 63) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6081) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1985) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 63))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 63) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 63) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 15) & 15))) && ((((((int)threadIdx.x) * 74) + 15) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 63) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 63) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 15) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + ((((int)threadIdx.x) * 74) >> 6)) < 95) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + ((((int)threadIdx.x) * 74) >> 4)) < 380) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6080) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1984) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 64))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 74) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 74) & 63) >> 4)) < 15)) && (1 <= ((((int)threadIdx.x) * 74) & 15))) && (((((int)threadIdx.x) * 74) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + (((((int)threadIdx.x) * 74) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 74) & 63) >> 4) * 14)) + ((((int)threadIdx.x) * 74) & 15)) + 181))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 65) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 65) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6079) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1983) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 65))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 1) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 1) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 1) & 15))) && ((((((int)threadIdx.x) * 74) + 1) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 65) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 1) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 1) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 66) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 66) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6078) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1982) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 66))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 2) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 2) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 2) & 15))) && ((((((int)threadIdx.x) * 74) + 2) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 66) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 2) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 2) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 67) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 67) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6077) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1981) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 67))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 3) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 3) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 3) & 15))) && ((((((int)threadIdx.x) * 74) + 3) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 67) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 3) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 3) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 68) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 68) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6076) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1980) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 68))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 4) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 4) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 4) & 15))) && ((((((int)threadIdx.x) * 74) + 4) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 68) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 4) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 4) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 69) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 69) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6075) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1979) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 69))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 5) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 5) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 5) & 15))) && ((((((int)threadIdx.x) * 74) + 5) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 69) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 5) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 5) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 70) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 70) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6074) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1978) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 70))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 6) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 6) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 6) & 15))) && ((((((int)threadIdx.x) * 74) + 6) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 70) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 6) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 6) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 71) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 71) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6073) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1977) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 71))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 7) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 7) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 7) & 15))) && ((((((int)threadIdx.x) * 74) + 7) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 71) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 7) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 7) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 72) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 72) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6072) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1976) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 72))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 8) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 8) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 8) & 15))) && ((((((int)threadIdx.x) * 74) + 8) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 72) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 8) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 8) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((((int)threadIdx.x) * 74) + 73) >> 6)) < 96) {
      if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) + (((((int)threadIdx.x) * 74) + 73) >> 4)) < 384) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) < 6071) {
          if (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 74)) < 1975) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.x) * 74)) + 73))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 9) & 63) >> 4))) && (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 74) + 9) & 63) >> 4)) < 15)) && (1 <= (((((int)threadIdx.x) * 74) + 9) & 15))) && ((((((int)threadIdx.x) * 74) + 9) & 15) < 15)) ? data[(((((((((rc_outer * 18816) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 3136)) + ((((((int)threadIdx.x) * 74) + 73) >> 6) * 196)) + (((int)blockIdx.y) * 28)) + (((((((int)threadIdx.x) * 74) + 9) & 63) >> 4) * 14)) + (((((int)threadIdx.x) * 74) + 9) & 15)) - 15))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    kernel_shared[((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)))] = kernel[((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 1))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 1))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 2))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 2))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 3))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 3))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 4))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 4))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 5))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 5))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 6))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 6))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 7))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 7))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 8))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 8))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 9))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 9))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 10))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 10))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 11))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 11))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 12))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 12))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 13))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 13))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 14))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 14))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 15))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 15))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 16))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 16))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 17))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 17))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 18))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 18))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 19))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 19))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 20))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 20))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 21))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 21))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 22))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 22))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 23))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 23))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 24))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 24))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 25))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 25))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 26))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 26))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 27))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 27))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 28))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 28))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 29))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 29))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 30))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 30))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 31))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 31))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 32))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 32))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 33))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 33))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 34))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 34))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 35))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 35))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 36))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 36))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 37))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 37))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 38))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 38))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 39))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 39))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 40))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 40))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 41))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 41))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 42))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 42))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 43))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 43))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 44))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 44))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 45))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 45))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 46))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 46))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 47))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 47))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 48))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 48))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 49))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 49))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 50))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 50))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 51))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 51))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 52))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 52))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 53))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 53))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 54))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 54))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 55))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 55))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 56))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 56))];
    kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 57))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 57))];
    if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 62) + 58) / 864)) + ((int)threadIdx.y)) < 6) {
      if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((((int)threadIdx.x) * 62) + 58) / 9)) < 576) {
        if ((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + (((((int)threadIdx.x) * 62) + 58) / 3)) < 1728) {
          if ((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) < 5126) {
            if (((((int)threadIdx.y) * 864) + (((int)threadIdx.x) * 62)) < 1670) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 58))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 58))];
              }
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 62) + 59) / 864)) + ((int)threadIdx.y)) < 6) {
      if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((((int)threadIdx.x) * 62) + 59) / 9)) < 576) {
        if ((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + (((((int)threadIdx.x) * 62) + 59) / 3)) < 1728) {
          if ((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) < 5125) {
            if (((((int)threadIdx.y) * 864) + (((int)threadIdx.x) * 62)) < 1669) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 59))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 59))];
              }
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 62) + 60) / 864)) + ((int)threadIdx.y)) < 6) {
      if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((((int)threadIdx.x) * 62) + 60) / 9)) < 576) {
        if ((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + ((((int)threadIdx.x) * 62) / 3)) < 1708) {
          if ((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) < 5124) {
            if (((((int)threadIdx.y) * 864) + (((int)threadIdx.x) * 62)) < 1668) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 60))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 60))];
              }
            }
          }
        }
      }
    }
    if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 62) + 61) / 864)) + ((int)threadIdx.y)) < 6) {
      if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((((int)threadIdx.x) * 62) + 61) / 9)) < 576) {
        if ((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + (((((int)threadIdx.x) * 62) + 61) / 3)) < 1728) {
          if ((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) < 5123) {
            if (((((int)threadIdx.y) * 864) + (((int)threadIdx.x) * 62)) < 1667) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1728) + (((int)threadIdx.y) * 864)) + (((int)threadIdx.x) * 62)) + 61))] = kernel[(((((((((int)blockIdx.z) * 10368) + (((int)threadIdx.z) * 3456)) + (((int)threadIdx.y) * 1728)) + (rc_outer * 864)) + (((int)threadIdx.x) * 62)) + 61))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 48; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 64))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 9))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 864))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 873))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 65))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 1))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 10))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 865))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 874))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 66))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 2))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 11))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 866))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 875))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 80))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 3))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 12))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 867))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 876))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 81))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 4))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 13))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 868))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 877))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 82))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 5))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 14))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 869))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 878))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 96))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 6))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 15))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 870))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 879))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 97))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 7))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 16))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 871))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 880))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 128) + (((int)threadIdx.y) * 16)) + ((int)threadIdx.x)) + 98))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 8))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 17))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 872))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 1728) + (rc_inner_outer * 18)) + 881))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    }
  }
  compute[((((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 392)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 392)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 196))] = compute_local[(1)];
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,7,16);

                dim3 block(14,2,3);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


