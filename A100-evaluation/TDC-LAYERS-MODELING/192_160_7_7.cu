#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 2
#define TC 16
#define C 192
#define N 160
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[1512];
  __shared__ float kernel_shared[288];
  float pad_temp_shared_local[6];
  float kernel_shared_local[6];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = ((((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) && (1 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 8))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 8))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) && (1 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 8))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 3))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = (((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 7))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 7))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 7))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 1))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 4))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = ((((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 6))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 6))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 6))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 2))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 5))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
  }
  compute[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,1,40);

                dim3 block(7,7,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


