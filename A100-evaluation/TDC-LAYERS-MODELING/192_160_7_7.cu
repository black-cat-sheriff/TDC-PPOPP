#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 2
#define TC 16
#define C 192
#define N 160
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[432];
  __shared__ float kernel_shared[1440];
  float pad_temp_shared_local[72];
  float kernel_shared_local[72];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
      __syncthreads();
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 432) {
        pad_temp_shared[(((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)))] = (((((1 <= (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9)) && ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) / 9) * 49)) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 431) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 430) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 429) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 428) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 427) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 426) {
        if (((int)threadIdx.y) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6))] = (((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) < 8)) && (1 <= (((int)blockIdx.x) + rx_outer))) && ((((int)blockIdx.x) + rx_outer) < 8)) ? data[(((((((rc_outer * 2352) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) * 7)) + ((int)blockIdx.x)) + rx_outer) - 8))] : 0.000000e+00f);
        }
      }
      kernel_shared[(((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)))] = kernel[((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 1))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 3))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 2))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 6))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 3))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 9))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 4))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 12))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 5))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 15))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 6))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 18))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 7))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 21))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 8))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 24))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 9))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 27))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 10))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 30))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 11))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 33))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 12))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 36))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 13))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 39))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 14))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 42))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 15))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 45))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 16))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 48))];
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 17))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 51))];
      if (((((((int)threadIdx.y) * 7) + 6) / 48) + ((int)threadIdx.z)) < 10) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 7)) < 474) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) < 1422) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 18))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 54))];
            }
          }
        }
      }
      if (((((((int)threadIdx.y) * 7) + 6) / 48) + ((int)threadIdx.z)) < 10) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 7)) < 474) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) < 1421) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 19))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 57))];
            }
          }
        }
      }
      if (((((((int)threadIdx.y) * 7) + 6) / 48) + ((int)threadIdx.z)) < 10) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 7)) < 474) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) < 1420) {
            if (((int)threadIdx.y) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 21)) + 20))] = kernel[(((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 1728)) + (rc_outer * 432)) + (((int)threadIdx.y) * 63)) + rx_outer) + 60))];
            }
          }
        }
      }
      __syncthreads();
      pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.y))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 9))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 10))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 11))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 18))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 19))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.y) + 20))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.y) + 27))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.y) + 28))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.y) + 29))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.y) + 36))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.y) + 37))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.y) + 38))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.y) + 45))];
      pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.y) + 46))];
      pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.y) + 47))];
      pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.y) + 54))];
      pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.y) + 55))];
      pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.y) + 56))];
      pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.y) + 63))];
      pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.y) + 64))];
      pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.y) + 65))];
      pad_temp_shared_local[(24)] = pad_temp_shared[((((int)threadIdx.y) + 72))];
      pad_temp_shared_local[(25)] = pad_temp_shared[((((int)threadIdx.y) + 73))];
      pad_temp_shared_local[(26)] = pad_temp_shared[((((int)threadIdx.y) + 74))];
      pad_temp_shared_local[(27)] = pad_temp_shared[((((int)threadIdx.y) + 81))];
      pad_temp_shared_local[(28)] = pad_temp_shared[((((int)threadIdx.y) + 82))];
      pad_temp_shared_local[(29)] = pad_temp_shared[((((int)threadIdx.y) + 83))];
      pad_temp_shared_local[(30)] = pad_temp_shared[((((int)threadIdx.y) + 90))];
      pad_temp_shared_local[(31)] = pad_temp_shared[((((int)threadIdx.y) + 91))];
      pad_temp_shared_local[(32)] = pad_temp_shared[((((int)threadIdx.y) + 92))];
      pad_temp_shared_local[(33)] = pad_temp_shared[((((int)threadIdx.y) + 99))];
      pad_temp_shared_local[(34)] = pad_temp_shared[((((int)threadIdx.y) + 100))];
      pad_temp_shared_local[(35)] = pad_temp_shared[((((int)threadIdx.y) + 101))];
      pad_temp_shared_local[(36)] = pad_temp_shared[((((int)threadIdx.y) + 108))];
      pad_temp_shared_local[(37)] = pad_temp_shared[((((int)threadIdx.y) + 109))];
      pad_temp_shared_local[(38)] = pad_temp_shared[((((int)threadIdx.y) + 110))];
      pad_temp_shared_local[(39)] = pad_temp_shared[((((int)threadIdx.y) + 117))];
      pad_temp_shared_local[(40)] = pad_temp_shared[((((int)threadIdx.y) + 118))];
      pad_temp_shared_local[(41)] = pad_temp_shared[((((int)threadIdx.y) + 119))];
      pad_temp_shared_local[(42)] = pad_temp_shared[((((int)threadIdx.y) + 126))];
      pad_temp_shared_local[(43)] = pad_temp_shared[((((int)threadIdx.y) + 127))];
      pad_temp_shared_local[(44)] = pad_temp_shared[((((int)threadIdx.y) + 128))];
      pad_temp_shared_local[(45)] = pad_temp_shared[((((int)threadIdx.y) + 135))];
      pad_temp_shared_local[(46)] = pad_temp_shared[((((int)threadIdx.y) + 136))];
      pad_temp_shared_local[(47)] = pad_temp_shared[((((int)threadIdx.y) + 137))];
      pad_temp_shared_local[(48)] = pad_temp_shared[((((int)threadIdx.y) + 144))];
      pad_temp_shared_local[(49)] = pad_temp_shared[((((int)threadIdx.y) + 145))];
      pad_temp_shared_local[(50)] = pad_temp_shared[((((int)threadIdx.y) + 146))];
      pad_temp_shared_local[(51)] = pad_temp_shared[((((int)threadIdx.y) + 153))];
      pad_temp_shared_local[(52)] = pad_temp_shared[((((int)threadIdx.y) + 154))];
      pad_temp_shared_local[(53)] = pad_temp_shared[((((int)threadIdx.y) + 155))];
      pad_temp_shared_local[(54)] = pad_temp_shared[((((int)threadIdx.y) + 162))];
      pad_temp_shared_local[(55)] = pad_temp_shared[((((int)threadIdx.y) + 163))];
      pad_temp_shared_local[(56)] = pad_temp_shared[((((int)threadIdx.y) + 164))];
      pad_temp_shared_local[(57)] = pad_temp_shared[((((int)threadIdx.y) + 171))];
      pad_temp_shared_local[(58)] = pad_temp_shared[((((int)threadIdx.y) + 172))];
      pad_temp_shared_local[(59)] = pad_temp_shared[((((int)threadIdx.y) + 173))];
      pad_temp_shared_local[(60)] = pad_temp_shared[((((int)threadIdx.y) + 180))];
      pad_temp_shared_local[(61)] = pad_temp_shared[((((int)threadIdx.y) + 181))];
      pad_temp_shared_local[(62)] = pad_temp_shared[((((int)threadIdx.y) + 182))];
      pad_temp_shared_local[(63)] = pad_temp_shared[((((int)threadIdx.y) + 189))];
      pad_temp_shared_local[(64)] = pad_temp_shared[((((int)threadIdx.y) + 190))];
      pad_temp_shared_local[(65)] = pad_temp_shared[((((int)threadIdx.y) + 191))];
      pad_temp_shared_local[(66)] = pad_temp_shared[((((int)threadIdx.y) + 198))];
      pad_temp_shared_local[(67)] = pad_temp_shared[((((int)threadIdx.y) + 199))];
      pad_temp_shared_local[(68)] = pad_temp_shared[((((int)threadIdx.y) + 200))];
      pad_temp_shared_local[(69)] = pad_temp_shared[((((int)threadIdx.y) + 207))];
      pad_temp_shared_local[(70)] = pad_temp_shared[((((int)threadIdx.y) + 208))];
      pad_temp_shared_local[(71)] = pad_temp_shared[((((int)threadIdx.y) + 209))];
      kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 144))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 1))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 2))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 3))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 4))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 5))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 144) + 6))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 144) + 7))];
      kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 144) + 8))];
      kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 144) + 9))];
      kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 144) + 10))];
      kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 144) + 11))];
      kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 144) + 12))];
      kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 144) + 13))];
      kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 144) + 14))];
      kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 144) + 15))];
      kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 144) + 16))];
      kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 144) + 17))];
      kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 144) + 18))];
      kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 144) + 19))];
      kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 144) + 20))];
      kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 144) + 21))];
      kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 144) + 22))];
      kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 144) + 23))];
      kernel_shared_local[(24)] = kernel_shared[(((((int)threadIdx.z) * 144) + 24))];
      kernel_shared_local[(25)] = kernel_shared[(((((int)threadIdx.z) * 144) + 25))];
      kernel_shared_local[(26)] = kernel_shared[(((((int)threadIdx.z) * 144) + 26))];
      kernel_shared_local[(27)] = kernel_shared[(((((int)threadIdx.z) * 144) + 27))];
      kernel_shared_local[(28)] = kernel_shared[(((((int)threadIdx.z) * 144) + 28))];
      kernel_shared_local[(29)] = kernel_shared[(((((int)threadIdx.z) * 144) + 29))];
      kernel_shared_local[(30)] = kernel_shared[(((((int)threadIdx.z) * 144) + 30))];
      kernel_shared_local[(31)] = kernel_shared[(((((int)threadIdx.z) * 144) + 31))];
      kernel_shared_local[(32)] = kernel_shared[(((((int)threadIdx.z) * 144) + 32))];
      kernel_shared_local[(33)] = kernel_shared[(((((int)threadIdx.z) * 144) + 33))];
      kernel_shared_local[(34)] = kernel_shared[(((((int)threadIdx.z) * 144) + 34))];
      kernel_shared_local[(35)] = kernel_shared[(((((int)threadIdx.z) * 144) + 35))];
      kernel_shared_local[(36)] = kernel_shared[(((((int)threadIdx.z) * 144) + 36))];
      kernel_shared_local[(37)] = kernel_shared[(((((int)threadIdx.z) * 144) + 37))];
      kernel_shared_local[(38)] = kernel_shared[(((((int)threadIdx.z) * 144) + 38))];
      kernel_shared_local[(39)] = kernel_shared[(((((int)threadIdx.z) * 144) + 39))];
      kernel_shared_local[(40)] = kernel_shared[(((((int)threadIdx.z) * 144) + 40))];
      kernel_shared_local[(41)] = kernel_shared[(((((int)threadIdx.z) * 144) + 41))];
      kernel_shared_local[(42)] = kernel_shared[(((((int)threadIdx.z) * 144) + 42))];
      kernel_shared_local[(43)] = kernel_shared[(((((int)threadIdx.z) * 144) + 43))];
      kernel_shared_local[(44)] = kernel_shared[(((((int)threadIdx.z) * 144) + 44))];
      kernel_shared_local[(45)] = kernel_shared[(((((int)threadIdx.z) * 144) + 45))];
      kernel_shared_local[(46)] = kernel_shared[(((((int)threadIdx.z) * 144) + 46))];
      kernel_shared_local[(47)] = kernel_shared[(((((int)threadIdx.z) * 144) + 47))];
      kernel_shared_local[(48)] = kernel_shared[(((((int)threadIdx.z) * 144) + 48))];
      kernel_shared_local[(49)] = kernel_shared[(((((int)threadIdx.z) * 144) + 49))];
      kernel_shared_local[(50)] = kernel_shared[(((((int)threadIdx.z) * 144) + 50))];
      kernel_shared_local[(51)] = kernel_shared[(((((int)threadIdx.z) * 144) + 51))];
      kernel_shared_local[(52)] = kernel_shared[(((((int)threadIdx.z) * 144) + 52))];
      kernel_shared_local[(53)] = kernel_shared[(((((int)threadIdx.z) * 144) + 53))];
      kernel_shared_local[(54)] = kernel_shared[(((((int)threadIdx.z) * 144) + 54))];
      kernel_shared_local[(55)] = kernel_shared[(((((int)threadIdx.z) * 144) + 55))];
      kernel_shared_local[(56)] = kernel_shared[(((((int)threadIdx.z) * 144) + 56))];
      kernel_shared_local[(57)] = kernel_shared[(((((int)threadIdx.z) * 144) + 57))];
      kernel_shared_local[(58)] = kernel_shared[(((((int)threadIdx.z) * 144) + 58))];
      kernel_shared_local[(59)] = kernel_shared[(((((int)threadIdx.z) * 144) + 59))];
      kernel_shared_local[(60)] = kernel_shared[(((((int)threadIdx.z) * 144) + 60))];
      kernel_shared_local[(61)] = kernel_shared[(((((int)threadIdx.z) * 144) + 61))];
      kernel_shared_local[(62)] = kernel_shared[(((((int)threadIdx.z) * 144) + 62))];
      kernel_shared_local[(63)] = kernel_shared[(((((int)threadIdx.z) * 144) + 63))];
      kernel_shared_local[(64)] = kernel_shared[(((((int)threadIdx.z) * 144) + 64))];
      kernel_shared_local[(65)] = kernel_shared[(((((int)threadIdx.z) * 144) + 65))];
      kernel_shared_local[(66)] = kernel_shared[(((((int)threadIdx.z) * 144) + 66))];
      kernel_shared_local[(67)] = kernel_shared[(((((int)threadIdx.z) * 144) + 67))];
      kernel_shared_local[(68)] = kernel_shared[(((((int)threadIdx.z) * 144) + 68))];
      kernel_shared_local[(69)] = kernel_shared[(((((int)threadIdx.z) * 144) + 69))];
      kernel_shared_local[(70)] = kernel_shared[(((((int)threadIdx.z) * 144) + 70))];
      kernel_shared_local[(71)] = kernel_shared[(((((int)threadIdx.z) * 144) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(24)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(25)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(26)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(27)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(28)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(29)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(30)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(31)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(32)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(33)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(34)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(35)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(36)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(37)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(38)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(39)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(40)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(41)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(42)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(43)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(44)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(45)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(46)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(47)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(48)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(49)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(50)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(51)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(52)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(53)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(54)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(55)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(56)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(57)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(58)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(59)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(60)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(61)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(62)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(63)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(64)] * kernel_shared_local[(64)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(65)] * kernel_shared_local[(65)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(66)] * kernel_shared_local[(66)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(67)] * kernel_shared_local[(67)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(68)] * kernel_shared_local[(68)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(69)] * kernel_shared_local[(69)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(70)] * kernel_shared_local[(70)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(71)] * kernel_shared_local[(71)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 216))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 217))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 218))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 225))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 226))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 227))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 234))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 235))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.y) + 236))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.y) + 243))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.y) + 244))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.y) + 245))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.y) + 252))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.y) + 253))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.y) + 254))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.y) + 261))];
      pad_temp_shared_local[(16)] = pad_temp_shared[((((int)threadIdx.y) + 262))];
      pad_temp_shared_local[(17)] = pad_temp_shared[((((int)threadIdx.y) + 263))];
      pad_temp_shared_local[(18)] = pad_temp_shared[((((int)threadIdx.y) + 270))];
      pad_temp_shared_local[(19)] = pad_temp_shared[((((int)threadIdx.y) + 271))];
      pad_temp_shared_local[(20)] = pad_temp_shared[((((int)threadIdx.y) + 272))];
      pad_temp_shared_local[(21)] = pad_temp_shared[((((int)threadIdx.y) + 279))];
      pad_temp_shared_local[(22)] = pad_temp_shared[((((int)threadIdx.y) + 280))];
      pad_temp_shared_local[(23)] = pad_temp_shared[((((int)threadIdx.y) + 281))];
      pad_temp_shared_local[(24)] = pad_temp_shared[((((int)threadIdx.y) + 288))];
      pad_temp_shared_local[(25)] = pad_temp_shared[((((int)threadIdx.y) + 289))];
      pad_temp_shared_local[(26)] = pad_temp_shared[((((int)threadIdx.y) + 290))];
      pad_temp_shared_local[(27)] = pad_temp_shared[((((int)threadIdx.y) + 297))];
      pad_temp_shared_local[(28)] = pad_temp_shared[((((int)threadIdx.y) + 298))];
      pad_temp_shared_local[(29)] = pad_temp_shared[((((int)threadIdx.y) + 299))];
      pad_temp_shared_local[(30)] = pad_temp_shared[((((int)threadIdx.y) + 306))];
      pad_temp_shared_local[(31)] = pad_temp_shared[((((int)threadIdx.y) + 307))];
      pad_temp_shared_local[(32)] = pad_temp_shared[((((int)threadIdx.y) + 308))];
      pad_temp_shared_local[(33)] = pad_temp_shared[((((int)threadIdx.y) + 315))];
      pad_temp_shared_local[(34)] = pad_temp_shared[((((int)threadIdx.y) + 316))];
      pad_temp_shared_local[(35)] = pad_temp_shared[((((int)threadIdx.y) + 317))];
      pad_temp_shared_local[(36)] = pad_temp_shared[((((int)threadIdx.y) + 324))];
      pad_temp_shared_local[(37)] = pad_temp_shared[((((int)threadIdx.y) + 325))];
      pad_temp_shared_local[(38)] = pad_temp_shared[((((int)threadIdx.y) + 326))];
      pad_temp_shared_local[(39)] = pad_temp_shared[((((int)threadIdx.y) + 333))];
      pad_temp_shared_local[(40)] = pad_temp_shared[((((int)threadIdx.y) + 334))];
      pad_temp_shared_local[(41)] = pad_temp_shared[((((int)threadIdx.y) + 335))];
      pad_temp_shared_local[(42)] = pad_temp_shared[((((int)threadIdx.y) + 342))];
      pad_temp_shared_local[(43)] = pad_temp_shared[((((int)threadIdx.y) + 343))];
      pad_temp_shared_local[(44)] = pad_temp_shared[((((int)threadIdx.y) + 344))];
      pad_temp_shared_local[(45)] = pad_temp_shared[((((int)threadIdx.y) + 351))];
      pad_temp_shared_local[(46)] = pad_temp_shared[((((int)threadIdx.y) + 352))];
      pad_temp_shared_local[(47)] = pad_temp_shared[((((int)threadIdx.y) + 353))];
      pad_temp_shared_local[(48)] = pad_temp_shared[((((int)threadIdx.y) + 360))];
      pad_temp_shared_local[(49)] = pad_temp_shared[((((int)threadIdx.y) + 361))];
      pad_temp_shared_local[(50)] = pad_temp_shared[((((int)threadIdx.y) + 362))];
      pad_temp_shared_local[(51)] = pad_temp_shared[((((int)threadIdx.y) + 369))];
      pad_temp_shared_local[(52)] = pad_temp_shared[((((int)threadIdx.y) + 370))];
      pad_temp_shared_local[(53)] = pad_temp_shared[((((int)threadIdx.y) + 371))];
      pad_temp_shared_local[(54)] = pad_temp_shared[((((int)threadIdx.y) + 378))];
      pad_temp_shared_local[(55)] = pad_temp_shared[((((int)threadIdx.y) + 379))];
      pad_temp_shared_local[(56)] = pad_temp_shared[((((int)threadIdx.y) + 380))];
      pad_temp_shared_local[(57)] = pad_temp_shared[((((int)threadIdx.y) + 387))];
      pad_temp_shared_local[(58)] = pad_temp_shared[((((int)threadIdx.y) + 388))];
      pad_temp_shared_local[(59)] = pad_temp_shared[((((int)threadIdx.y) + 389))];
      pad_temp_shared_local[(60)] = pad_temp_shared[((((int)threadIdx.y) + 396))];
      pad_temp_shared_local[(61)] = pad_temp_shared[((((int)threadIdx.y) + 397))];
      pad_temp_shared_local[(62)] = pad_temp_shared[((((int)threadIdx.y) + 398))];
      pad_temp_shared_local[(63)] = pad_temp_shared[((((int)threadIdx.y) + 405))];
      pad_temp_shared_local[(64)] = pad_temp_shared[((((int)threadIdx.y) + 406))];
      pad_temp_shared_local[(65)] = pad_temp_shared[((((int)threadIdx.y) + 407))];
      pad_temp_shared_local[(66)] = pad_temp_shared[((((int)threadIdx.y) + 414))];
      pad_temp_shared_local[(67)] = pad_temp_shared[((((int)threadIdx.y) + 415))];
      pad_temp_shared_local[(68)] = pad_temp_shared[((((int)threadIdx.y) + 416))];
      pad_temp_shared_local[(69)] = pad_temp_shared[((((int)threadIdx.y) + 423))];
      pad_temp_shared_local[(70)] = pad_temp_shared[((((int)threadIdx.y) + 424))];
      pad_temp_shared_local[(71)] = pad_temp_shared[((((int)threadIdx.y) + 425))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 72))];
      kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 73))];
      kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 74))];
      kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 75))];
      kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 76))];
      kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 77))];
      kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 144) + 78))];
      kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 144) + 79))];
      kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 144) + 80))];
      kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 144) + 81))];
      kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 144) + 82))];
      kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 144) + 83))];
      kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 144) + 84))];
      kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 144) + 85))];
      kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 144) + 86))];
      kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 144) + 87))];
      kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 144) + 88))];
      kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 144) + 89))];
      kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 144) + 90))];
      kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 144) + 91))];
      kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 144) + 92))];
      kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 144) + 93))];
      kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 144) + 94))];
      kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 144) + 95))];
      kernel_shared_local[(24)] = kernel_shared[(((((int)threadIdx.z) * 144) + 96))];
      kernel_shared_local[(25)] = kernel_shared[(((((int)threadIdx.z) * 144) + 97))];
      kernel_shared_local[(26)] = kernel_shared[(((((int)threadIdx.z) * 144) + 98))];
      kernel_shared_local[(27)] = kernel_shared[(((((int)threadIdx.z) * 144) + 99))];
      kernel_shared_local[(28)] = kernel_shared[(((((int)threadIdx.z) * 144) + 100))];
      kernel_shared_local[(29)] = kernel_shared[(((((int)threadIdx.z) * 144) + 101))];
      kernel_shared_local[(30)] = kernel_shared[(((((int)threadIdx.z) * 144) + 102))];
      kernel_shared_local[(31)] = kernel_shared[(((((int)threadIdx.z) * 144) + 103))];
      kernel_shared_local[(32)] = kernel_shared[(((((int)threadIdx.z) * 144) + 104))];
      kernel_shared_local[(33)] = kernel_shared[(((((int)threadIdx.z) * 144) + 105))];
      kernel_shared_local[(34)] = kernel_shared[(((((int)threadIdx.z) * 144) + 106))];
      kernel_shared_local[(35)] = kernel_shared[(((((int)threadIdx.z) * 144) + 107))];
      kernel_shared_local[(36)] = kernel_shared[(((((int)threadIdx.z) * 144) + 108))];
      kernel_shared_local[(37)] = kernel_shared[(((((int)threadIdx.z) * 144) + 109))];
      kernel_shared_local[(38)] = kernel_shared[(((((int)threadIdx.z) * 144) + 110))];
      kernel_shared_local[(39)] = kernel_shared[(((((int)threadIdx.z) * 144) + 111))];
      kernel_shared_local[(40)] = kernel_shared[(((((int)threadIdx.z) * 144) + 112))];
      kernel_shared_local[(41)] = kernel_shared[(((((int)threadIdx.z) * 144) + 113))];
      kernel_shared_local[(42)] = kernel_shared[(((((int)threadIdx.z) * 144) + 114))];
      kernel_shared_local[(43)] = kernel_shared[(((((int)threadIdx.z) * 144) + 115))];
      kernel_shared_local[(44)] = kernel_shared[(((((int)threadIdx.z) * 144) + 116))];
      kernel_shared_local[(45)] = kernel_shared[(((((int)threadIdx.z) * 144) + 117))];
      kernel_shared_local[(46)] = kernel_shared[(((((int)threadIdx.z) * 144) + 118))];
      kernel_shared_local[(47)] = kernel_shared[(((((int)threadIdx.z) * 144) + 119))];
      kernel_shared_local[(48)] = kernel_shared[(((((int)threadIdx.z) * 144) + 120))];
      kernel_shared_local[(49)] = kernel_shared[(((((int)threadIdx.z) * 144) + 121))];
      kernel_shared_local[(50)] = kernel_shared[(((((int)threadIdx.z) * 144) + 122))];
      kernel_shared_local[(51)] = kernel_shared[(((((int)threadIdx.z) * 144) + 123))];
      kernel_shared_local[(52)] = kernel_shared[(((((int)threadIdx.z) * 144) + 124))];
      kernel_shared_local[(53)] = kernel_shared[(((((int)threadIdx.z) * 144) + 125))];
      kernel_shared_local[(54)] = kernel_shared[(((((int)threadIdx.z) * 144) + 126))];
      kernel_shared_local[(55)] = kernel_shared[(((((int)threadIdx.z) * 144) + 127))];
      kernel_shared_local[(56)] = kernel_shared[(((((int)threadIdx.z) * 144) + 128))];
      kernel_shared_local[(57)] = kernel_shared[(((((int)threadIdx.z) * 144) + 129))];
      kernel_shared_local[(58)] = kernel_shared[(((((int)threadIdx.z) * 144) + 130))];
      kernel_shared_local[(59)] = kernel_shared[(((((int)threadIdx.z) * 144) + 131))];
      kernel_shared_local[(60)] = kernel_shared[(((((int)threadIdx.z) * 144) + 132))];
      kernel_shared_local[(61)] = kernel_shared[(((((int)threadIdx.z) * 144) + 133))];
      kernel_shared_local[(62)] = kernel_shared[(((((int)threadIdx.z) * 144) + 134))];
      kernel_shared_local[(63)] = kernel_shared[(((((int)threadIdx.z) * 144) + 135))];
      kernel_shared_local[(64)] = kernel_shared[(((((int)threadIdx.z) * 144) + 136))];
      kernel_shared_local[(65)] = kernel_shared[(((((int)threadIdx.z) * 144) + 137))];
      kernel_shared_local[(66)] = kernel_shared[(((((int)threadIdx.z) * 144) + 138))];
      kernel_shared_local[(67)] = kernel_shared[(((((int)threadIdx.z) * 144) + 139))];
      kernel_shared_local[(68)] = kernel_shared[(((((int)threadIdx.z) * 144) + 140))];
      kernel_shared_local[(69)] = kernel_shared[(((((int)threadIdx.z) * 144) + 141))];
      kernel_shared_local[(70)] = kernel_shared[(((((int)threadIdx.z) * 144) + 142))];
      kernel_shared_local[(71)] = kernel_shared[(((((int)threadIdx.z) * 144) + 143))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(24)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(25)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(26)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(27)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(28)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(29)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(30)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(31)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(32)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(33)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(34)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(35)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(36)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(37)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(38)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(39)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(40)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(41)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(42)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(43)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(44)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(45)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(46)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(47)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(48)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(49)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(50)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(51)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(52)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(53)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(54)] * kernel_shared_local[(54)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(55)] * kernel_shared_local[(55)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(56)] * kernel_shared_local[(56)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(57)] * kernel_shared_local[(57)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(58)] * kernel_shared_local[(58)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(59)] * kernel_shared_local[(59)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(60)] * kernel_shared_local[(60)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(61)] * kernel_shared_local[(61)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(62)] * kernel_shared_local[(62)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(63)] * kernel_shared_local[(63)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(64)] * kernel_shared_local[(64)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(65)] * kernel_shared_local[(65)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(66)] * kernel_shared_local[(66)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(67)] * kernel_shared_local[(67)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(68)] * kernel_shared_local[(68)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(69)] * kernel_shared_local[(69)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(70)] * kernel_shared_local[(70)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(71)] * kernel_shared_local[(71)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 490) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)blockIdx.x)))] = compute_local[(0)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(7,1,16);

                dim3 block(1,7,10);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


