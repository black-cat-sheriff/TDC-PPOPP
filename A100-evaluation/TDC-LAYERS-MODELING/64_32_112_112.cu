#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 4
#define TC 16
#define C 64
#define N 32
#define H 112
#define W 112

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[1920];
  __shared__ float kernel_shared[768];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + ((((int)threadIdx.x) * 9) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((((int)threadIdx.x) * 9) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1920) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 480) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)))] = (((((1 <= (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 9) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 9) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + ((((int)threadIdx.x) * 9) % 30)))) && (((((int)blockIdx.x) * 28) + ((((int)threadIdx.x) * 9) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + (((((int)threadIdx.x) * 9) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + ((((((int)threadIdx.x) * 9) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + ((((int)threadIdx.x) * 9) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 1) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 1) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1919) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 479) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 1))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 1) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 1) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 1) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 1) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 1) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 1) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 1) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 2) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 2) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1918) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 478) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 2))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 2) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 2) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 2) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 2) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 2) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 2) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 2) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 3) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 3) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1917) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 477) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 3))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 3) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 3) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 3) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 3) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 3) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 3) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 3) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 4) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 4) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1916) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 476) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 4))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 4) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 4) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 4) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 4) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 4) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 4) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 4) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 5) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 5) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1915) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 475) {
              if (((int)threadIdx.x) < 27) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 5))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 5) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 5) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 5) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 5) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 5) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 5) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 5) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 6) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 6) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1914) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 474) {
              if (((int)threadIdx.x) < 26) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 6))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 6) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 6) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 6) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 6) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 6) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 6) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 6) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 7) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 7) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1913) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 473) {
              if (((int)threadIdx.x) < 26) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 7))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 7) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 7) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 7) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 7) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 7) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 7) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 7) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 9) + 8) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 9) + 8) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) < 1912) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 9)) < 472) {
              if (((int)threadIdx.x) < 26) {
                pad_temp_shared[(((((((int)threadIdx.z) * 480) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 9)) + 8))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 8) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 9) + 8) % 120) / 30)) + ry_outer) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 8) % 30)))) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.x) * 9) + 8) % 30)) < 113)) ? data[(((((((((((rc_outer * 200704) + (((int)threadIdx.z) * 50176)) + (((int)threadIdx.y) * 25088)) + ((((((int)threadIdx.x) * 9) + 8) / 120) * 12544)) + (((int)blockIdx.y) * 448)) + (((((((int)threadIdx.x) * 9) + 8) % 120) / 30) * 112)) + (ry_outer * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.x) * 9) + 8) % 30)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((int)threadIdx.x) / 12)) < 16) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + ((((int)threadIdx.x) * 4) / 3)) < 256) {
          if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) < 768) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 4)) < 192) {
              if (((int)threadIdx.x) < 24) {
                kernel_shared[((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 1152)) + ((((int)threadIdx.x) / 12) * 576)) + (rc_outer * 144)) + ((((((int)threadIdx.x) % 12) * 4) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 4) % 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 4) + 1) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((((int)threadIdx.x) * 4) + 1) / 3)) < 256) {
          if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) < 767) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 4)) < 191) {
              if (((int)threadIdx.x) < 24) {
                kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) + 1))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 1152)) + ((((((int)threadIdx.x) * 4) + 1) / 48) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 4) + 1) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 4) + 1) % 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 4) + 2) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + (((((int)threadIdx.x) * 4) + 2) / 3)) < 256) {
          if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) < 766) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 4)) < 190) {
              if (((int)threadIdx.x) < 24) {
                kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) + 2))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 1152)) + ((((((int)threadIdx.x) * 4) + 2) / 48) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 4) + 2) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 4) + 2) % 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 4) + 3) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) + ((((int)threadIdx.x) * 4) / 3)) < 255) {
          if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) < 765) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 4)) < 189) {
              if (((int)threadIdx.x) < 24) {
                kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 4)) + 3))] = kernel[(((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 1152)) + ((((((int)threadIdx.x) * 4) + 3) / 48) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 4) + 3) % 48) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 4) % 3)))];
              }
            }
          }
        }
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 16; ++rc_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)) + 60))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)) + 1))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)) + 61))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 120) + (((int)threadIdx.y) * 30)) + ((int)threadIdx.x)) + 62))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 1))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 2))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 48))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 49))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 50))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 96))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 97))];
        kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 98))];
        kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 144))];
        kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 145))];
        kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 3)) + 146))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(9)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(10)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(11)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      }
    }
  }
  compute[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 224))] = compute_local[(4)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 12544))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 12768))] = compute_local[(5)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 25088))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 25312))] = compute_local[(6)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 37632))] = compute_local[(3)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 37856))] = compute_local[(7)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(4,28,2);

                dim3 block(28,2,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


