#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 32
#define N 32
#define H 14
#define W 14

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[576];
  __shared__ float kernel_shared[72];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 576) {
      pad_temp_shared[(((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)))] = (((((9 <= (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144)) && ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) < 15)) ? data[(((((((rc_outer * 784) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) / 144) * 196)) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 575) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 574) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 573) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 572) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 571) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 24) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 72) {
        if (((int)threadIdx.x) < 6) {
          kernel_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))] = kernel[((((((((int)blockIdx.z) * 576) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) / 12) * 288)) + (rc_outer * 36)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) % 12) * 3)) + (((int)threadIdx.x) % 3)))];
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 9) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 144))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 153))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 162))];
    kernel_shared_local[(0)] = kernel_shared[(0)];
    kernel_shared_local[(1)] = kernel_shared[(3)];
    kernel_shared_local[(2)] = kernel_shared[(6)];
    kernel_shared_local[(3)] = kernel_shared[(9)];
    kernel_shared_local[(4)] = kernel_shared[(12)];
    kernel_shared_local[(5)] = kernel_shared[(15)];
    kernel_shared_local[(6)] = kernel_shared[(36)];
    kernel_shared_local[(7)] = kernel_shared[(39)];
    kernel_shared_local[(8)] = kernel_shared[(42)];
    kernel_shared_local[(9)] = kernel_shared[(45)];
    kernel_shared_local[(10)] = kernel_shared[(48)];
    kernel_shared_local[(11)] = kernel_shared[(51)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 145))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 154))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 163))];
    kernel_shared_local[(0)] = kernel_shared[(1)];
    kernel_shared_local[(1)] = kernel_shared[(4)];
    kernel_shared_local[(2)] = kernel_shared[(7)];
    kernel_shared_local[(3)] = kernel_shared[(10)];
    kernel_shared_local[(4)] = kernel_shared[(13)];
    kernel_shared_local[(5)] = kernel_shared[(16)];
    kernel_shared_local[(6)] = kernel_shared[(37)];
    kernel_shared_local[(7)] = kernel_shared[(40)];
    kernel_shared_local[(8)] = kernel_shared[(43)];
    kernel_shared_local[(9)] = kernel_shared[(46)];
    kernel_shared_local[(10)] = kernel_shared[(49)];
    kernel_shared_local[(11)] = kernel_shared[(52)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 146))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 155))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 164))];
    kernel_shared_local[(0)] = kernel_shared[(2)];
    kernel_shared_local[(1)] = kernel_shared[(5)];
    kernel_shared_local[(2)] = kernel_shared[(8)];
    kernel_shared_local[(3)] = kernel_shared[(11)];
    kernel_shared_local[(4)] = kernel_shared[(14)];
    kernel_shared_local[(5)] = kernel_shared[(17)];
    kernel_shared_local[(6)] = kernel_shared[(38)];
    kernel_shared_local[(7)] = kernel_shared[(41)];
    kernel_shared_local[(8)] = kernel_shared[(44)];
    kernel_shared_local[(9)] = kernel_shared[(47)];
    kernel_shared_local[(10)] = kernel_shared[(50)];
    kernel_shared_local[(11)] = kernel_shared[(53)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 288))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 297))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 306))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 432))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 450))];
    kernel_shared_local[(0)] = kernel_shared[(18)];
    kernel_shared_local[(1)] = kernel_shared[(21)];
    kernel_shared_local[(2)] = kernel_shared[(24)];
    kernel_shared_local[(3)] = kernel_shared[(27)];
    kernel_shared_local[(4)] = kernel_shared[(30)];
    kernel_shared_local[(5)] = kernel_shared[(33)];
    kernel_shared_local[(6)] = kernel_shared[(54)];
    kernel_shared_local[(7)] = kernel_shared[(57)];
    kernel_shared_local[(8)] = kernel_shared[(60)];
    kernel_shared_local[(9)] = kernel_shared[(63)];
    kernel_shared_local[(10)] = kernel_shared[(66)];
    kernel_shared_local[(11)] = kernel_shared[(69)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 289))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 298))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 307))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 433))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 442))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 451))];
    kernel_shared_local[(0)] = kernel_shared[(19)];
    kernel_shared_local[(1)] = kernel_shared[(22)];
    kernel_shared_local[(2)] = kernel_shared[(25)];
    kernel_shared_local[(3)] = kernel_shared[(28)];
    kernel_shared_local[(4)] = kernel_shared[(31)];
    kernel_shared_local[(5)] = kernel_shared[(34)];
    kernel_shared_local[(6)] = kernel_shared[(55)];
    kernel_shared_local[(7)] = kernel_shared[(58)];
    kernel_shared_local[(8)] = kernel_shared[(61)];
    kernel_shared_local[(9)] = kernel_shared[(64)];
    kernel_shared_local[(10)] = kernel_shared[(67)];
    kernel_shared_local[(11)] = kernel_shared[(70)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 290))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 299))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 308))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 434))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 443))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 452))];
    kernel_shared_local[(0)] = kernel_shared[(20)];
    kernel_shared_local[(1)] = kernel_shared[(23)];
    kernel_shared_local[(2)] = kernel_shared[(26)];
    kernel_shared_local[(3)] = kernel_shared[(29)];
    kernel_shared_local[(4)] = kernel_shared[(32)];
    kernel_shared_local[(5)] = kernel_shared[(35)];
    kernel_shared_local[(6)] = kernel_shared[(56)];
    kernel_shared_local[(7)] = kernel_shared[(59)];
    kernel_shared_local[(8)] = kernel_shared[(62)];
    kernel_shared_local[(9)] = kernel_shared[(65)];
    kernel_shared_local[(10)] = kernel_shared[(68)];
    kernel_shared_local[(11)] = kernel_shared[(71)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(1)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(2,1,16);
    dim3 block(7,14,1);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


