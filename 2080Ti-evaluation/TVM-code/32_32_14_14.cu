#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 32
#define N 32
#define H 14
#define W 14

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[216];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[3];
  float kernel_shared_local[3];
  compute_local[(0)] = 0.000000e+00f;
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) - 15))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) - 15))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) - 15))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) - 15))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 1))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 2))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 3))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 4))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 5))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 6))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 7))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 8))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 9))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 10))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 1553))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 1553))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 1553))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 1553))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 72))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 73))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 74))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 75))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 76))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 77))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 78))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 79))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 80))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 81))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 82))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 3121))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 3121))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 3121))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 3121))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 144))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 145))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 146))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 147))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 148))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 149))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 150))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 151))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 152))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 153))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 154))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 4689))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 4689))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 4689))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 4689))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 216))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 217))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 218))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 219))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 220))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 221))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 222))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 223))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 224))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 225))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 226))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  compute[((((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(2,14,4);
    dim3 block(7,1,8);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


