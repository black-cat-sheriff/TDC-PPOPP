#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 32
#define N 32
#define H 28
#define W 28

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[864];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[16];
  float kernel_shared_local[4];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((int)threadIdx.x) * 8) / 9))) && (((((int)blockIdx.y) * 4) + ((((int)threadIdx.x) * 8) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 8) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 8) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((((int)threadIdx.x) * 8) / 9) * 28)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 8) % 9)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 1) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 1) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 1) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 1) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 1) % 9)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 2) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 2) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 2) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 2) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 2) % 9)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 3) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 3) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 3) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 3) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 3) % 9)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 4) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 4) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 4) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 4) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 4) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 4) % 9)) - 29))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 5) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 5) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 5) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 5) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 5) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 5) % 9)) - 29))] : 0.000000e+00f);
    if (((((((int)threadIdx.x) * 8) + 6) / 54) + ((int)threadIdx.z)) < 16) {
      if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 6) / 9)) < 96) {
        if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 858) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 6) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 6) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 6) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 6) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 6) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 6) % 9)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 8) + 7) / 54) + ((int)threadIdx.z)) < 16) {
      if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 7) / 9)) < 96) {
        if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 857) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 7) / 9))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 8) + 7) / 9)) < 29)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 7) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 8) + 7) % 9)) < 29)) ? data[((((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 8) + 7) / 9) * 28)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 8) + 7) % 9)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)))] = kernel[(((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 1))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 2))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 4))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 5))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 7))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 8))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 10))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 11))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 13))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 14))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 15))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 16))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 16))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 17))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 17))];
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 16) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 254) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 762) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 2286) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 18))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 16) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 254) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 762) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 2285) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 19))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 19))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 16) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 254) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 762) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 2284) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 20))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 20))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 144))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 9))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 18))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 27))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 19))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 28))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 11))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 3))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 12))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 21))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 30))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 4))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 22))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 31))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 5))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 14))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 23))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 32))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 15))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 24))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 33))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 7))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 25))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 34))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 8))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 17))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 216))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 225))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 234))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 243))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 270))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 279))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 288))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 297))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 324))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 333))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 342))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 351))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 387))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 396))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 405))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 45))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 54))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 63))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 226))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 235))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 244))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 271))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 289))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 298))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 325))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 334))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 352))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 379))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 388))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 397))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 37))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 46))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 55))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 64))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 218))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 227))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 236))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 272))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 281))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 290))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 299))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 326))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 335))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 344))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 353))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 380))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 389))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 398))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 407))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 38))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 47))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 225))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 234))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 243))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 279))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 288))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 297))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 306))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 333))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 342))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 351))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 360))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 387))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 396))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 405))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 414))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 39))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 48))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 57))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 66))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 226))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 235))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 244))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 253))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 289))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 298))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 307))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 334))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 352))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 361))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 388))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 397))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 415))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 40))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 58))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 67))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 227))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 236))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 254))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 281))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 290))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 299))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 335))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 344))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 353))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 362))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 389))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 398))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 407))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 416))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 41))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 50))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 59))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 68))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 234))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 243))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 261))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 288))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 297))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 306))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 342))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 351))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 360))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 369))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 396))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 405))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 414))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 423))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 51))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 60))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 235))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 244))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 253))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 262))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 289))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 298))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 307))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 316))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 352))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 361))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 370))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 397))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 415))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 424))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 43))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 61))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 236))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 254))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 263))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 290))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 299))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 317))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 344))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 353))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 362))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 398))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 407))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 416))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 425))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 44))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 53))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 432))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 450))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 459))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 486))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 495))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 504))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 513))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 540))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 549))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 558))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 567))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 594))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 603))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 612))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 621))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 81))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 90))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 99))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 433))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 442))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 451))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 460))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 487))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 496))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 505))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 514))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 541))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 550))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 559))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 568))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 595))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 604))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 613))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 622))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 73))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 82))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 91))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 100))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 434))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 443))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 452))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 461))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 488))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 506))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 515))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 542))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 551))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 560))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 569))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 596))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 605))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 614))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 623))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 74))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 83))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 92))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 101))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 441))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 450))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 459))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 468))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 495))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 504))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 513))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 522))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 549))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 558))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 567))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 576))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 603))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 612))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 621))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 630))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 75))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 84))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 93))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 102))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 442))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 451))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 460))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 496))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 505))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 514))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 523))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 550))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 559))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 568))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 577))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 604))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 613))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 622))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 631))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 76))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 85))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 94))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 103))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 443))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 452))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 461))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 470))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 497))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 506))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 515))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 524))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 551))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 560))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 569))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 578))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 605))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 614))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 623))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 632))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 77))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 86))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 95))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 104))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 450))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 459))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 468))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 477))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 504))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 513))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 522))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 531))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 558))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 567))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 576))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 585))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 612))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 621))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 630))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 639))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 78))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 87))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 96))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 105))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 451))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 460))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 469))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 478))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 505))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 514))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 523))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 532))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 559))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 568))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 577))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 586))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 613))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 622))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 631))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 640))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 79))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 88))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 97))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 106))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 452))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 461))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 470))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 479))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 506))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 515))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 524))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 533))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 560))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 569))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 578))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 587))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 614))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 623))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 632))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 641))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 80))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 89))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 98))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 107))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 648))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 657))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 666))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 675))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 702))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 711))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 720))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 729))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 756))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 765))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 774))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 783))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 810))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 819))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 828))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 837))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 108))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 117))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 126))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 135))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 649))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 658))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 667))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 676))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 703))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 712))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 721))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 730))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 757))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 766))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 775))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 784))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 811))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 820))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 829))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 838))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 109))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 118))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 127))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 136))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 650))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 659))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 668))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 677))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 704))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 713))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 722))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 731))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 758))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 767))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 776))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 785))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 812))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 821))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 830))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 839))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 110))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 119))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 128))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 137))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 657))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 666))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 675))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 684))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 711))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 720))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 729))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 738))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 765))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 774))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 783))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 792))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 819))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 828))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 837))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 846))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 111))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 120))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 129))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 138))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 658))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 667))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 676))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 685))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 712))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 721))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 730))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 739))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 766))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 775))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 784))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 793))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 820))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 829))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 838))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 847))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 112))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 121))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 130))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 139))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 659))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 668))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 677))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 686))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 713))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 722))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 731))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 740))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 767))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 776))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 785))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 794))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 821))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 830))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 839))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 848))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 113))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 122))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 131))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 140))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 666))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 675))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 684))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 720))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 729))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 738))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 747))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 774))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 783))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 792))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 801))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 828))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 837))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 846))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 855))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 114))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 123))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 132))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 141))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 667))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 676))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 685))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 694))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 721))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 730))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 739))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 748))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 775))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 784))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 793))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 802))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 829))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 838))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 847))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 856))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 115))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 124))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 133))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 142))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 668))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 677))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 686))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 695))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 722))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 731))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 740))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 749))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 776))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 785))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 794))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 803))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((int)threadIdx.x) + 830))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((int)threadIdx.x) + 839))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((int)threadIdx.x) + 848))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((int)threadIdx.x) + 857))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 116))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 125))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 134))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 143))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
  }
  compute[((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 28))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 56))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 84))] = compute_local[(3)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(4,7,2);
    dim3 block(7,1,16);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


