#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 192
#define N 160
#define H 7
#define W 7

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[216];
  __shared__ float kernel_shared[360];
  float pad_temp_shared_local[8];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 216) {
      pad_temp_shared[(((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)))] = ((((1 <= (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9)) && ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) / 9) * 49)) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 215) {
      pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 214) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 213) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 212) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 211) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 210) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) < 8)) && (1 <= ((int)blockIdx.x))) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) * 7)) + ((int)blockIdx.x)) - 8))] : 0.000000e+00f);
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)))] = kernel[(((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 15))];
    if (((((((int)threadIdx.y) * 11) + 6) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 118) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 354) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 18))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 7) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 7) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 353) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 21))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 8) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 8) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 352) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 24))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 9) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 117) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 351) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 27))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 10) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 10) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 350) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 30))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.y))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 27))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 36))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 45))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 54))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 63))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 28))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 37))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 46))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 55))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 64))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 29))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 38))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 47))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 56))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 65))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 72))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 81))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 90))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 99))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 108))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 117))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 135))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 73))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 82))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 100))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 109))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 118))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 127))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 136))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 74))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 83))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 92))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 101))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 110))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 128))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 137))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 144))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 153))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 162))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 171))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 180))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 189))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 198))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 145))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 154))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 163))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 172))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 181))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 190))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 199))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 208))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 146))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 155))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 164))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 173))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 182))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 191))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 200))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 209))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    __syncthreads();
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 216) {
      pad_temp_shared[(((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)))] = (((1 <= (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9)) && ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) < 8)) ? data[((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) / 9) * 49)) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 215) {
      pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 214) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 213) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 212) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 211) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 210) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6))] = (((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) < 8)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) * 7)) + ((int)blockIdx.x)) - 7))] : 0.000000e+00f);
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 16))];
    if (((((((int)threadIdx.y) * 11) + 6) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 118) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 354) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 19))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 7) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 7) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 353) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 22))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 8) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 8) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 352) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 25))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 9) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 117) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 351) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 28))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 10) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 10) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 350) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 31))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.y))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 27))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 36))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 45))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 54))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 63))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 28))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 37))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 46))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 55))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 64))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 29))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 38))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 47))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 56))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 65))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 72))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 81))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 90))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 99))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 108))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 117))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 135))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 73))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 82))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 100))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 109))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 118))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 127))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 136))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 74))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 83))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 92))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 101))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 110))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 128))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 137))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 144))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 153))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 162))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 171))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 180))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 189))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 198))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 145))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 154))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 163))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 172))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 181))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 190))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 199))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 208))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 146))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 155))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 164))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 173))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 182))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 191))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 200))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 209))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    __syncthreads();
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 216) {
      pad_temp_shared[(((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)))] = ((((1 <= (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9)) && ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) / 9) * 49)) + ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 215) {
      pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 1) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 214) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 2) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 213) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 3) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 212) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 4) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 211) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 5) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
      }
    }
    if (((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) < 210) {
      if (((int)threadIdx.y) < 6) {
        pad_temp_shared[((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6))] = ((((1 <= ((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9)) && (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) < 8)) && (((int)blockIdx.x) < 6)) ? data[((((((rc_outer * 1176) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) / 9) * 49)) + (((((((int)threadIdx.z) * 44) + (((int)threadIdx.y) * 7)) + 6) % 9) * 7)) + ((int)blockIdx.x)) - 6))] : 0.000000e+00f);
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 17))];
    if (((((((int)threadIdx.y) * 11) + 6) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 118) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 354) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 20))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 7) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 7) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 353) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 23))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 8) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 8) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 352) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 26))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 9) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.y) * 11) / 3)) < 117) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 351) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 29))];
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + 10) / 72) + ((int)threadIdx.z)) < 5) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + 10) / 3)) < 120) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) < 350) {
          if (((int)threadIdx.y) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + 32))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.y))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 27))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 36))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 45))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 54))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 63))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 28))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 37))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 46))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 55))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 64))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 29))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 38))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 47))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 56))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 65))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 72))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 81))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 90))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 99))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 108))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 117))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 126))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 135))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 73))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 82))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 100))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 109))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 118))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 127))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 136))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 74))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 83))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 92))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 101))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 110))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 119))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 128))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 137))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 144))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 153))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 162))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 171))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 180))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 189))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 198))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 145))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 154))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 163))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 172))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 181))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 190))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 199))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 208))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.y) + 146))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.y) + 155))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.y) + 164))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.y) + 173))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.y) + 182))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.y) + 191))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.y) + 200))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.y) + 209))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
  }
  compute[(((((((int)blockIdx.z) * 245) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)blockIdx.x)))] = compute_local[(0)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(7,1,32);
    dim3 block(1,7,5);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


