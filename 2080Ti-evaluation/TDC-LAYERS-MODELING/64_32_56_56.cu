#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 6
#define TC 16
#define C 64
#define N 32
#define H 56
#define W 56

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[28];
  __shared__ float pad_temp_shared[1856];
  __shared__ float kernel_shared[1536];
  float pad_temp_shared_local[14];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
      __syncthreads();
      pad_temp_shared[((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)))] = (((((2 <= (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) % 116)) && ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) / 116) * 3136)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 1))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 2))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 2) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 2) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 2) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 2) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 3))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 3) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 3) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 3) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 3) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 4))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 4) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 4) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 4) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 4) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 5))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 5) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 5) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 5) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 5) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 6))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 6) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 6) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 6) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 6) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 7))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 7) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 7) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 7) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 7) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 8))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 8) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 8) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 8) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 8) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 9))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 9) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 9) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 9) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 9) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 10))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 10) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 10) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 10) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 10) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 11))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 11) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 11) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 11) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 11) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 12))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 12) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 12) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 12) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 12) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 13))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 13) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 13) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 13) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 13) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 1) & 1)) - 57))] : 0.000000e+00f);
      if (((((int)threadIdx.z) * 2) + ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 14) / 116)) < 16) {
        if (((((int)threadIdx.z) * 116) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) >> 1)) < 921) {
          if ((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) < 1842) {
            if (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) < 218) {
              if (((int)threadIdx.x) < 1) {
                pad_temp_shared[(((((((int)threadIdx.z) * 232) + (((int)threadIdx.y) * 29)) + (((int)threadIdx.x) * 15)) + 14))] = (((((2 <= ((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 14) % 116)) && (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 14) % 116) < 114)) && (1 <= (((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)))) && ((((((int)blockIdx.x) * 2) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) < 57)) ? data[(((((((((rc_outer * 50176) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 14) / 116) * 3136)) + ((((((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) + 14) % 116) >> 1) * 56)) + (((int)blockIdx.x) * 2)) + rx_outer) + (((((int)threadIdx.y) * 29) + (((int)threadIdx.x) * 15)) & 1)) - 57))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      kernel_shared[((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)))] = kernel[((((((((int)threadIdx.z) * 2304) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) >> 4) * 576)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) & 15) * 9)) + rx_outer))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 1))] = kernel[(((((((((int)threadIdx.z) * 2304) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) >> 4) * 576)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) & 15) * 9)) + rx_outer) + 3))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 2))] = kernel[(((((((((int)threadIdx.z) * 2304) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) >> 4) * 576)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) & 15) * 9)) + rx_outer) + 6))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 3))] = kernel[((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) & 15) * 9)) + rx_outer))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 4))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) & 15) * 9)) + rx_outer) + 3))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 5))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 1) & 15) * 9)) + rx_outer) + 6))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 6))] = kernel[((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) & 15) * 9)) + rx_outer))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 7))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) & 15) * 9)) + rx_outer) + 3))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 8))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 2) & 15) * 9)) + rx_outer) + 6))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 9))] = kernel[((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) & 15) * 9)) + rx_outer))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 10))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) & 15) * 9)) + rx_outer) + 3))];
      kernel_shared[(((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + 11))] = kernel[(((((((((int)threadIdx.z) * 2304) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 4)) + 3) & 15) * 9)) + rx_outer) + 6))];
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 4))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 6))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 8))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 10))];
        pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 12))];
        pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 116))];
        pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 118))];
        pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 120))];
        pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 122))];
        pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 124))];
        pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 126))];
        pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 128))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 3))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 48))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 51))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 96))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 99))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 144))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 147))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(2)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(4)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(4)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(6)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(6)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(3)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(5)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(7)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(7)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(7)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 4))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 6))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 8))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 10))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 12))];
        pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 14))];
        pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 118))];
        pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 120))];
        pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 122))];
        pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 124))];
        pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 126))];
        pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 128))];
        pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 130))];
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 1))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 4))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 49))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 52))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 97))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 100))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 145))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 148))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(2)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(4)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(4)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(6)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(6)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(3)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(5)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(7)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(7)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(7)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 4))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 6))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 8))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 10))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 12))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 14))];
        pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 16))];
        pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 120))];
        pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 122))];
        pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 124))];
        pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 126))];
        pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 128))];
        pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 130))];
        pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 232) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) + 132))];
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 2))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 5))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 50))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 53))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 98))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 101))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 146))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 192) + (rc_inner_outer * 6)) + 149))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(2)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(4)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(4)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(6)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(6)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(3)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(3)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
        compute_local[(16)] = (compute_local[(16)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
        compute_local[(17)] = (compute_local[(17)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
        compute_local[(18)] = (compute_local[(18)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
        compute_local[(19)] = (compute_local[(19)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(5)]));
        compute_local[(20)] = (compute_local[(20)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(5)]));
        compute_local[(21)] = (compute_local[(21)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(22)] = (compute_local[(22)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(7)]));
        compute_local[(23)] = (compute_local[(23)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
        compute_local[(24)] = (compute_local[(24)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
        compute_local[(25)] = (compute_local[(25)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(7)]));
        compute_local[(26)] = (compute_local[(26)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(7)]));
        compute_local[(27)] = (compute_local[(27)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
      }
    }
  }
  compute[(((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 56))] = compute_local[(1)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 112))] = compute_local[(2)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 168))] = compute_local[(3)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 224))] = compute_local[(4)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 280))] = compute_local[(5)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 336))] = compute_local[(6)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3136))] = compute_local[(7)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3192))] = compute_local[(8)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3248))] = compute_local[(9)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3304))] = compute_local[(10)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3360))] = compute_local[(11)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3416))] = compute_local[(12)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 3472))] = compute_local[(13)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6272))] = compute_local[(14)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6328))] = compute_local[(15)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6384))] = compute_local[(16)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6440))] = compute_local[(17)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6496))] = compute_local[(18)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6552))] = compute_local[(19)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 6608))] = compute_local[(20)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9408))] = compute_local[(21)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9464))] = compute_local[(22)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9520))] = compute_local[(23)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9576))] = compute_local[(24)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9632))] = compute_local[(25)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9688))] = compute_local[(26)];
  compute[((((((((int)threadIdx.z) * 12544) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 9744))] = compute_local[(27)];
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(4-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(5-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(6-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(7-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(0-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(1-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(2-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(3-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(4-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(5-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(6-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(7-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(0-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(1-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(2-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(3-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(4-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(5-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(6-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(7-s)] += result;
				}
			}
		break;
		case 24:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(0-s)] += result;
				}
			}
		break;
		case 25:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(1-s)] += result;
				}
			}
		break;
		case 26:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(2-s)] += result;
				}
			}
		break;
		case 27:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(3-s)] += result;
				}
			}
		break;
		case 28:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(4-s)] += result;
				}
			}
		break;
		case 29:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(5-s)] += result;
				}
			}
		break;
		case 30:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(6-s)] += result;
				}
			}
		break;
		case 31:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(7-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(28,1,1);

        dim3 block(2,8,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
                      cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
                      cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;
    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
    time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


