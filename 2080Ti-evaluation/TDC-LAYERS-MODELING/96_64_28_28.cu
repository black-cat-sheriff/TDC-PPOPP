#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 6
#define TC 16
#define C 96
#define N 64
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[3072];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[2];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)))] = (((((1 <= (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 55) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 55) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + (((((int)threadIdx.x) * 55) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 55) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 55) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 1))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 1) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 1) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 1) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 1) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 1) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 2))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 2) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 2) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 2) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 2) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 2) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 3))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 3) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 3) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 3) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 3) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 3) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 4))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 4) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 4) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 4) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 4) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 4) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 5))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 5) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 5) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 5) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 5) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 5) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 6))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 6) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 6) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 6) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 6) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 6) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 7))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 7) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 7) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 7) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 7) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 7) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 8))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 8) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 8) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 8) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 8) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 8) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 9))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 9) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 9) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 9) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 9) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 9) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 10))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 10) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 10) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 10) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 10) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 10) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 11))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 11) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 11) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 11) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 11) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 11) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 12))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 12) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 12) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 12) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 12) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 12) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 13))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 13) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 13) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 13) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 13) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 13) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 14))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 14) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 14) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 14) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 14) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 14) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 15))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 15) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 15) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 15) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 15) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 15) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 16))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 16) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 16) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 16) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 16) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 55) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 17))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 17) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 17) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 17) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 17) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 1) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 18))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 18) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 18) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 18) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 18) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 2) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 19))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 19) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 19) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 19) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 19) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 3) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 20))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 20) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 20) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 20) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 20) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 4) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 21))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 21) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 21) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 21) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 21) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 5) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 22))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 22) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 22) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 22) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 22) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 6) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 23))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 23) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 23) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 23) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 23) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 7) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 24))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 24) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 24) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 24) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 24) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 8) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 25))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 25) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 25) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 25) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 25) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 9) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 26))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 26) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 26) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 26) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 26) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 10) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 27))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 27) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 27) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 27) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 27) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 11) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 28))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 28) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 28) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 28) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 28) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 12) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 29))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 29) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 29) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 29) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 29) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 13) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 30))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 30) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 30) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 30) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 30) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 14) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 31))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 31) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 31) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 31) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 31) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 15) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 32))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 32) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 32) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 32) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 32) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 55) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 33))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 33) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 33) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 33) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 33) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 1) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 34))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 34) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 34) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 34) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 34) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 2) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 35))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 35) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 35) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 35) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 35) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 3) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 36))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 36) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 36) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 36) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 36) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 4) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 37))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 37) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 37) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 37) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 37) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 5) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 38))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 38) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 38) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 38) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 38) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 6) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 39))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 39) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 39) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 7) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 39) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 39) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 7) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 40))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 40) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 40) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 8) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 40) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 40) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 8) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 41))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 41) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 41) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 9) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 41) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 41) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 9) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 42))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 42) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 42) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 10) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 42) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 42) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 10) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 43))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 43) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 43) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 11) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 43) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 43) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 11) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 44))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 44) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 44) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 12) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 44) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 44) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 12) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 45))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 45) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 45) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 13) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 45) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 45) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 13) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 46))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 46) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 46) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 14) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 46) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 46) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 14) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 47))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 47) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 47) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 15) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 47) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 47) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 15) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 48))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 48) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 48) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)))) && (((((int)blockIdx.x) * 14) + ((((int)threadIdx.x) * 55) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 48) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 48) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + ((((int)threadIdx.x) * 55) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 49))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 49) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 49) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 1) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 49) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 49) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 1) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 50))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 50) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 50) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 2) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 50) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 50) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 2) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 51))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 51) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 51) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 3) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 51) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 51) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 3) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 52))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 52) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 52) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 4) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 52) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 52) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 4) & 15)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 53))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 53) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 53) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 5) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 53) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 53) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 5) & 15)) - 29))] : 0.000000e+00f);
      if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 6)) + (((((int)threadIdx.x) * 55) + 54) >> 6)) < 48) {
        if ((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 24)) + (((((int)threadIdx.x) * 55) + 54) >> 4)) < 192) {
          if ((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) < 3018) {
            if (((((int)threadIdx.y) * 384) + (((int)threadIdx.x) * 55)) < 1482) {
              if (((int)threadIdx.x) < 6) {
                pad_temp_shared[(((((((int)threadIdx.z) * 1536) + (((int)threadIdx.y) * 384)) + (((int)threadIdx.x) * 55)) + 54))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 54) & 63) >> 4)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 55) + 54) & 63) >> 4)) + ry_outer) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 55) + 6) & 15)) < 29)) ? data[(((((((((((rc_outer * 37632) + (((int)threadIdx.z) * 18816)) + (((int)threadIdx.y) * 4704)) + ((((((int)threadIdx.x) * 55) + 54) >> 6) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 55) + 54) & 63) >> 4) * 28)) + (ry_outer * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 55) + 6) & 15)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      kernel_shared[((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + (((((int)threadIdx.x) * 14) / 48) * 864)) + (rc_outer * 432)) + (((((int)threadIdx.x) * 14) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 1))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + (((((int)threadIdx.x) * 14) / 48) * 864)) + (rc_outer * 432)) + (((((int)threadIdx.x) * 14) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 2))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + (((((int)threadIdx.x) * 14) / 48) * 864)) + (rc_outer * 432)) + (((((int)threadIdx.x) * 14) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 3))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 4))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 5))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 1) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 1) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 6))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 7))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 8))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 2) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 2) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 9))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 10))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 11))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 3) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 3) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 12))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 13))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 14))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 4) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 4) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 15))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 16))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 17))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 5) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 5) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 18))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 19))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 20))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 6) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 6) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 21))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 22))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 23))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 7) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 7) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 24))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 25))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 26))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 8) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 8) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 27))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 28))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 29))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 9) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 9) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 30))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 31))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 32))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 10) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 10) % 48) * 9)) + (ry_outer * 3)) + 2))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 33))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 9)) + (ry_outer * 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 34))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 9)) + (ry_outer * 3)) + 1))];
      kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 35))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 11) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 11) % 48) * 9)) + (ry_outer * 3)) + 2))];
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 756) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2268) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1116) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 36))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 9)) + (ry_outer * 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 756) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2267) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1115) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 37))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 9)) + (ry_outer * 3)) + 1))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 12) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 756) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2266) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1114) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 38))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 12) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 12) % 48) * 9)) + (ry_outer * 3)) + 2))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 755) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2265) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1113) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 39))] = kernel[((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 9)) + (ry_outer * 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 755) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2264) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1112) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 40))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 9)) + (ry_outer * 3)) + 1))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 14) + 13) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 14)) < 755) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) < 2263) {
            if (((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 42)) < 1111) {
              if (((int)threadIdx.x) < 6) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 42)) + 41))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 14) + 13) / 48) * 864)) + (rc_outer * 432)) + ((((((int)threadIdx.x) * 14) + 13) % 48) * 9)) + (ry_outer * 3)) + 2))];
              }
            }
          }
        }
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 48; ++rc_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1152))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 144))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1296))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 288))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1440))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 432))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1584))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(1)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(5)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(7)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1153))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 145))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1297))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 289))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1441))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 433))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1585))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(1)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(5)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(7)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 2))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1154))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 146))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1298))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 290))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1442))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 434))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 3)) + 1586))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(1)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(5)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(7)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      }
    }
  }
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 6272))] = compute_local[(8)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 6273))] = compute_local[(9)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 784))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 7056))] = compute_local[(10)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 785))] = compute_local[(3)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 7057))] = compute_local[(11)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1568))] = compute_local[(4)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 7840))] = compute_local[(12)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1569))] = compute_local[(5)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 7841))] = compute_local[(13)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 2352))] = compute_local[(6)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 8624))] = compute_local[(14)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 2353))] = compute_local[(7)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 8625))] = compute_local[(15)];
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(4-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(5-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(6-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*6+(7-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(0-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(1-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(2-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(3-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(4-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(5-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(6-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*6+(7-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(0-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(1-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(2-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(3-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(4-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(5-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(6-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*6+(7-s)] += result;
				}
			}
		break;
		case 24:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(0-s)] += result;
				}
			}
		break;
		case 25:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(1-s)] += result;
				}
			}
		break;
		case 26:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(2-s)] += result;
				}
			}
		break;
		case 27:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(3-s)] += result;
				}
			}
		break;
		case 28:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(4-s)] += result;
				}
			}
		break;
		case 29:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(5-s)] += result;
				}
			}
		break;
		case 30:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(6-s)] += result;
				}
			}
		break;
		case 31:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*6+(7-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,7,4);

        dim3 block(7,4,2);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
                      cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
                      cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;
    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
    time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


