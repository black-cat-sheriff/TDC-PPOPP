#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 3
#define TC 16
#define C 32
#define N 32
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[4096];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[12];
  float kernel_shared_local[48];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)))] = (((((1 <= ((((int)blockIdx.y) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 5) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 6) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 7) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 8) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 9))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 9) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 10))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 10) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 11))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 11) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 12) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 13) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 14) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 15) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 16) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 16) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 16) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 16) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) & 15)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) >> 8)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) >> 4)) < 256) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) < 4079) {
          if (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) < 2031) {
            pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 17) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 1) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) >> 8)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) >> 4)) < 256) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) < 4078) {
          if (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) < 2030) {
            pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 18))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 18) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 2) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) >> 8)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) >> 4)) < 256) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) < 4077) {
          if (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) < 2029) {
            pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 19))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 19) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 3) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) >> 8)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) >> 4)) < 256) {
        if ((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) < 4076) {
          if (((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) < 2028) {
            if (((int)threadIdx.x) < 13) {
              pad_temp_shared[(((((((int)threadIdx.z) * 2048) + (((int)threadIdx.y) * 293)) + (((int)threadIdx.x) * 21)) + 20))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)) < 29)) ? data[(((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 20) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 293) + (((int)threadIdx.x) * 21)) + 4) & 15)) - 29))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    kernel_shared[((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 48) * 288)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) % 48) * 3)))];
    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 1))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 48) * 288)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) % 48) * 3)) + 1))];
    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 2))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 48) * 288)) + (rc_outer * 144)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) % 48) * 3)) + 2))];
    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 3))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) % 48) * 3)))];
    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 4))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) % 48) * 3)) + 1))];
    kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 5))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 1) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 3)) < 256) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 766) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2298) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1146) {
              if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 8)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 32) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 6))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) % 48) * 3)))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 3)) < 256) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 766) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2297) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1145) {
              if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 8)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 32) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 7))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) % 48) * 3)) + 1))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 3)) < 256) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 766) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2296) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1144) {
              if ((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 8)) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48)) < 32) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 8))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 2) % 48) * 3)) + 2))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + (((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 3)) < 255) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 765) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2295) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1143) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 9))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) % 48) * 3)))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + (((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 3)) < 255) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 765) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2294) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1142) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 10))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) % 48) * 3)) + 1))];
              }
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48)) < 16) {
      if (((((int)threadIdx.z) * 128) + (((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) / 3)) < 255) {
        if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 55)) + (((int)threadIdx.x) * 4)) < 765) {
          if ((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) < 2293) {
            if (((((int)threadIdx.y) * 165) + (((int)threadIdx.x) * 12)) < 1141) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 1152) + (((int)threadIdx.y) * 165)) + (((int)threadIdx.x) * 12)) + 11))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) / 48) * 288)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 55) + (((int)threadIdx.x) * 4)) + 3) % 48) * 3)) + 2))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 256))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 257))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 258))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 272))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 273))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 274))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)))];
      kernel_shared_local[(24)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1152))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1))];
      kernel_shared_local[(25)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1153))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 2))];
      kernel_shared_local[(26)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1154))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 9))];
      kernel_shared_local[(27)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1161))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 10))];
      kernel_shared_local[(28)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1162))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 11))];
      kernel_shared_local[(29)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1163))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 144))];
      kernel_shared_local[(30)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1296))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 145))];
      kernel_shared_local[(31)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1297))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 146))];
      kernel_shared_local[(32)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1298))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 153))];
      kernel_shared_local[(33)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1305))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 154))];
      kernel_shared_local[(34)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1306))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 155))];
      kernel_shared_local[(35)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1307))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 288))];
      kernel_shared_local[(36)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1440))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 289))];
      kernel_shared_local[(37)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1441))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 290))];
      kernel_shared_local[(38)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1442))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 297))];
      kernel_shared_local[(39)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1449))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 298))];
      kernel_shared_local[(40)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1450))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 299))];
      kernel_shared_local[(41)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1451))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 432))];
      kernel_shared_local[(42)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1584))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 433))];
      kernel_shared_local[(43)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1585))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 434))];
      kernel_shared_local[(44)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1586))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 441))];
      kernel_shared_local[(45)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1593))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 442))];
      kernel_shared_local[(46)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1594))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 443))];
      kernel_shared_local[(47)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1595))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(24)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(24)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(30)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(30)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(36)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(12)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(36)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(18)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(42)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(18)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(42)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(25)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(25)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(31)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(31)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(37)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(13)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(37)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(19)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(43)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(19)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(43)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(26)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(26)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(32)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(32)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(38)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(14)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(38)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(20)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(44)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(20)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(44)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(27)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(27)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(33)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(33)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(15)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(39)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(15)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(39)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(21)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(45)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(45)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(28)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(28)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(34)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(34)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(16)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(40)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(16)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(40)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(22)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(46)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(46)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(29)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(29)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(35)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(35)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(17)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(41)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(17)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(41)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(23)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(47)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(47)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 272))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 273))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 274))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 288))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 289))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 290))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 3))];
      kernel_shared_local[(24)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1155))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 4))];
      kernel_shared_local[(25)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1156))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 5))];
      kernel_shared_local[(26)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1157))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 12))];
      kernel_shared_local[(27)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1164))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 13))];
      kernel_shared_local[(28)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1165))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 14))];
      kernel_shared_local[(29)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1166))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 147))];
      kernel_shared_local[(30)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1299))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 148))];
      kernel_shared_local[(31)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1300))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 149))];
      kernel_shared_local[(32)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1301))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 156))];
      kernel_shared_local[(33)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1308))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 157))];
      kernel_shared_local[(34)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1309))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 158))];
      kernel_shared_local[(35)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1310))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 291))];
      kernel_shared_local[(36)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1443))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 292))];
      kernel_shared_local[(37)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1444))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 293))];
      kernel_shared_local[(38)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1445))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 300))];
      kernel_shared_local[(39)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1452))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 301))];
      kernel_shared_local[(40)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1453))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 302))];
      kernel_shared_local[(41)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1454))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 435))];
      kernel_shared_local[(42)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1587))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 436))];
      kernel_shared_local[(43)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1588))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 437))];
      kernel_shared_local[(44)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1589))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 444))];
      kernel_shared_local[(45)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1596))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 445))];
      kernel_shared_local[(46)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1597))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 446))];
      kernel_shared_local[(47)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1598))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(24)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(24)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(30)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(30)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(36)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(12)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(36)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(18)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(42)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(18)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(42)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(25)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(25)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(31)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(31)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(37)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(13)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(37)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(19)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(43)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(19)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(43)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(26)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(26)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(32)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(32)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(38)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(14)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(38)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(20)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(44)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(20)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(44)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(27)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(27)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(33)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(33)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(15)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(39)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(15)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(39)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(21)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(45)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(45)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(28)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(28)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(34)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(34)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(16)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(40)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(16)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(40)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(22)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(46)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(46)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(29)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(29)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(35)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(35)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(17)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(41)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(17)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(41)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(23)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(47)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(47)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 48))];
      pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 49))];
      pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 50))];
      pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 288))];
      pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 289))];
      pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 290))];
      pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 304))];
      pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 305))];
      pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 512) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 306))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 6))];
      kernel_shared_local[(24)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1158))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 7))];
      kernel_shared_local[(25)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1159))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 8))];
      kernel_shared_local[(26)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1160))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 15))];
      kernel_shared_local[(27)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1167))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 16))];
      kernel_shared_local[(28)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1168))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 17))];
      kernel_shared_local[(29)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1169))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 150))];
      kernel_shared_local[(30)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1302))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 151))];
      kernel_shared_local[(31)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1303))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 152))];
      kernel_shared_local[(32)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1304))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 159))];
      kernel_shared_local[(33)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1311))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 160))];
      kernel_shared_local[(34)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1312))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 161))];
      kernel_shared_local[(35)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1313))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 294))];
      kernel_shared_local[(36)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1446))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 295))];
      kernel_shared_local[(37)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1447))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 296))];
      kernel_shared_local[(38)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1448))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 303))];
      kernel_shared_local[(39)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1455))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 304))];
      kernel_shared_local[(40)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1456))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 305))];
      kernel_shared_local[(41)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1457))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 438))];
      kernel_shared_local[(42)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1590))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 439))];
      kernel_shared_local[(43)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1591))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 440))];
      kernel_shared_local[(44)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1592))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 447))];
      kernel_shared_local[(45)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1599))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 448))];
      kernel_shared_local[(46)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1600))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 449))];
      kernel_shared_local[(47)] = kernel_shared[((((((int)threadIdx.z) * 576) + (rc_inner_outer * 18)) + 1601))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(24)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(24)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(30)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(30)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(36)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(12)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(36)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(18)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(42)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(18)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(42)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(25)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(25)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(31)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(31)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(37)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(13)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(37)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(19)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(43)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(19)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(43)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(26)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(26)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(32)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(32)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(38)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(14)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(38)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(20)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(44)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(20)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(44)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(27)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(27)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(33)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(33)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(15)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(39)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(15)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(39)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(21)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(45)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(45)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(28)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(28)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(34)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(34)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(16)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(40)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(16)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(40)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(22)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(46)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(46)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(29)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(29)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(35)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(35)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(17)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(41)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(17)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(41)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(23)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(47)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(47)]));
    }
  }
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 6272))] = compute_local[(8)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 28))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 6300))] = compute_local[(9)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 784))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 7056))] = compute_local[(10)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 812))] = compute_local[(3)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 7084))] = compute_local[(11)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 1568))] = compute_local[(4)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 7840))] = compute_local[(12)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 1596))] = compute_local[(5)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 7868))] = compute_local[(13)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 2352))] = compute_local[(6)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 8624))] = compute_local[(14)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 2380))] = compute_local[(7)];
  compute[((((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)) + 8652))] = compute_local[(15)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,2,2);

        dim3 block(14,7,2);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


