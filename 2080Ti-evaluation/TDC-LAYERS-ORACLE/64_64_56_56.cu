#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 4
#define TW 4
#define TC 16
#define C 64
#define N 64
#define H 56
#define W 56

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[56];
  __shared__ float pad_temp_shared[960];
  __shared__ float kernel_shared[384];
  float pad_temp_shared_local[28];
  float kernel_shared_local[2];
  #pragma unroll
  for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
    compute_local[(xx_c_init)] = 0.000000e+00f;
    compute_local[((xx_c_init + 28))] = 0.000000e+00f;
    compute_local[((xx_c_init + 2))] = 0.000000e+00f;
    compute_local[((xx_c_init + 30))] = 0.000000e+00f;
    compute_local[((xx_c_init + 4))] = 0.000000e+00f;
    compute_local[((xx_c_init + 32))] = 0.000000e+00f;
    compute_local[((xx_c_init + 6))] = 0.000000e+00f;
    compute_local[((xx_c_init + 34))] = 0.000000e+00f;
    compute_local[((xx_c_init + 8))] = 0.000000e+00f;
    compute_local[((xx_c_init + 36))] = 0.000000e+00f;
    compute_local[((xx_c_init + 10))] = 0.000000e+00f;
    compute_local[((xx_c_init + 38))] = 0.000000e+00f;
    compute_local[((xx_c_init + 12))] = 0.000000e+00f;
    compute_local[((xx_c_init + 40))] = 0.000000e+00f;
    compute_local[((xx_c_init + 14))] = 0.000000e+00f;
    compute_local[((xx_c_init + 42))] = 0.000000e+00f;
    compute_local[((xx_c_init + 16))] = 0.000000e+00f;
    compute_local[((xx_c_init + 44))] = 0.000000e+00f;
    compute_local[((xx_c_init + 18))] = 0.000000e+00f;
    compute_local[((xx_c_init + 46))] = 0.000000e+00f;
    compute_local[((xx_c_init + 20))] = 0.000000e+00f;
    compute_local[((xx_c_init + 48))] = 0.000000e+00f;
    compute_local[((xx_c_init + 22))] = 0.000000e+00f;
    compute_local[((xx_c_init + 50))] = 0.000000e+00f;
    compute_local[((xx_c_init + 24))] = 0.000000e+00f;
    compute_local[((xx_c_init + 52))] = 0.000000e+00f;
    compute_local[((xx_c_init + 26))] = 0.000000e+00f;
    compute_local[((xx_c_init + 54))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 120; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        pad_temp_shared[((((((int)threadIdx.z) * 240) + (((int)threadIdx.y) * 120)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30)) + ry_outer) < 57)) && (1 <= ((((int)blockIdx.x) * 28) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)))) && (((((int)blockIdx.x) * 28) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)) < 57)) ? data[(((((((((((rc_outer * 50176) + (((int)threadIdx.z) * 12544)) + (((int)threadIdx.y) * 6272)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 60) * 3136)) + (((int)blockIdx.y) * 112)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30) * 56)) + (ry_outer * 56)) + (((int)blockIdx.x) * 28)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)) - 57))] : 0.000000e+00f);
      }
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
        kernel_shared[((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 48)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 144)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3) * 9)) + (ry_outer * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 16; ++rc_inner_outer) {
        #pragma unroll
        for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(ax3)] = pad_temp_shared[(((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer))];
            pad_temp_shared_local[((ax3 + 2))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 2))];
            pad_temp_shared_local[((ax3 + 4))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 4))];
            pad_temp_shared_local[((ax3 + 6))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 6))];
            pad_temp_shared_local[((ax3 + 8))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 8))];
            pad_temp_shared_local[((ax3 + 10))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 10))];
            pad_temp_shared_local[((ax3 + 12))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 12))];
            pad_temp_shared_local[((ax3 + 14))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 14))];
            pad_temp_shared_local[((ax3 + 16))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 16))];
            pad_temp_shared_local[((ax3 + 18))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 18))];
            pad_temp_shared_local[((ax3 + 20))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 20))];
            pad_temp_shared_local[((ax3 + 22))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 22))];
            pad_temp_shared_local[((ax3 + 24))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 24))];
            pad_temp_shared_local[((ax3 + 26))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 26))];
          }
          kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 3)) + rx_inner_outer))];
          kernel_shared_local[(1)] = kernel_shared[(((((((int)threadIdx.z) * 48) + (rc_inner_outer * 3)) + rx_inner_outer) + 192))];
          #pragma unroll
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute_local[(xx_c)] = (compute_local[(xx_c)] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 28))] = (compute_local[((xx_c + 28))] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 2))] = (compute_local[((xx_c + 2))] + (pad_temp_shared_local[((xx_c + 2))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 30))] = (compute_local[((xx_c + 30))] + (pad_temp_shared_local[((xx_c + 2))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 4))] = (compute_local[((xx_c + 4))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 32))] = (compute_local[((xx_c + 32))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 6))] = (compute_local[((xx_c + 6))] + (pad_temp_shared_local[((xx_c + 6))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 34))] = (compute_local[((xx_c + 34))] + (pad_temp_shared_local[((xx_c + 6))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 8))] = (compute_local[((xx_c + 8))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 36))] = (compute_local[((xx_c + 36))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 10))] = (compute_local[((xx_c + 10))] + (pad_temp_shared_local[((xx_c + 10))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 38))] = (compute_local[((xx_c + 38))] + (pad_temp_shared_local[((xx_c + 10))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 12))] = (compute_local[((xx_c + 12))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 40))] = (compute_local[((xx_c + 40))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 14))] = (compute_local[((xx_c + 14))] + (pad_temp_shared_local[((xx_c + 14))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 42))] = (compute_local[((xx_c + 42))] + (pad_temp_shared_local[((xx_c + 14))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 16))] = (compute_local[((xx_c + 16))] + (pad_temp_shared_local[((xx_c + 16))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 44))] = (compute_local[((xx_c + 44))] + (pad_temp_shared_local[((xx_c + 16))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 18))] = (compute_local[((xx_c + 18))] + (pad_temp_shared_local[((xx_c + 18))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 46))] = (compute_local[((xx_c + 46))] + (pad_temp_shared_local[((xx_c + 18))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 20))] = (compute_local[((xx_c + 20))] + (pad_temp_shared_local[((xx_c + 20))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 48))] = (compute_local[((xx_c + 48))] + (pad_temp_shared_local[((xx_c + 20))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 22))] = (compute_local[((xx_c + 22))] + (pad_temp_shared_local[((xx_c + 22))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 50))] = (compute_local[((xx_c + 50))] + (pad_temp_shared_local[((xx_c + 22))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 24))] = (compute_local[((xx_c + 24))] + (pad_temp_shared_local[((xx_c + 24))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 52))] = (compute_local[((xx_c + 52))] + (pad_temp_shared_local[((xx_c + 24))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 26))] = (compute_local[((xx_c + 26))] + (pad_temp_shared_local[((xx_c + 26))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 54))] = (compute_local[((xx_c + 54))] + (pad_temp_shared_local[((xx_c + 26))] * kernel_shared_local[(1)]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
    compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner))] = compute_local[(xx_inner_inner_inner)];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12544))] = compute_local[((xx_inner_inner_inner + 28))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 2))] = compute_local[((xx_inner_inner_inner + 2))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12546))] = compute_local[((xx_inner_inner_inner + 30))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 4))] = compute_local[((xx_inner_inner_inner + 4))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12548))] = compute_local[((xx_inner_inner_inner + 32))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 6))] = compute_local[((xx_inner_inner_inner + 6))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12550))] = compute_local[((xx_inner_inner_inner + 34))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 8))] = compute_local[((xx_inner_inner_inner + 8))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12552))] = compute_local[((xx_inner_inner_inner + 36))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 10))] = compute_local[((xx_inner_inner_inner + 10))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12554))] = compute_local[((xx_inner_inner_inner + 38))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12))] = compute_local[((xx_inner_inner_inner + 12))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12556))] = compute_local[((xx_inner_inner_inner + 40))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 14))] = compute_local[((xx_inner_inner_inner + 14))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12558))] = compute_local[((xx_inner_inner_inner + 42))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 16))] = compute_local[((xx_inner_inner_inner + 16))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12560))] = compute_local[((xx_inner_inner_inner + 44))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 18))] = compute_local[((xx_inner_inner_inner + 18))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12562))] = compute_local[((xx_inner_inner_inner + 46))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 20))] = compute_local[((xx_inner_inner_inner + 20))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12564))] = compute_local[((xx_inner_inner_inner + 48))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 22))] = compute_local[((xx_inner_inner_inner + 22))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12566))] = compute_local[((xx_inner_inner_inner + 50))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 24))] = compute_local[((xx_inner_inner_inner + 24))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12568))] = compute_local[((xx_inner_inner_inner + 52))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 26))] = compute_local[((xx_inner_inner_inner + 26))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12570))] = compute_local[((xx_inner_inner_inner + 54))];
  }
}





class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(4-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(5-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(0-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(1-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(2-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(3-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(4-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(5-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(0-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(1-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(2-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(3-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(4-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(5-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(0-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(1-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(2-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(3-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(4-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 0; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(5-s)] += result;
				}
			}
		break;
		case 24:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(0-s)] += result;
				}
			}
		break;
		case 25:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(1-s)] += result;
				}
			}
		break;
		case 26:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(2-s)] += result;
				}
			}
		break;
		case 27:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(3-s)] += result;
				}
			}
		break;
		case 28:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(4-s)] += result;
				}
			}
		break;
		case 29:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(4-r)*4+(5-s)] += result;
				}
			}
		break;
		case 30:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(0-s)] += result;
				}
			}
		break;
		case 31:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(1-s)] += result;
				}
			}
		break;
		case 32:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(2-s)] += result;
				}
			}
		break;
		case 33:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(3-s)] += result;
				}
			}
		break;
		case 34:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(4-s)] += result;
				}
			}
		break;
		case 35:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(5-r)*4+(5-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,28,8);

        dim3 block(1,2,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


