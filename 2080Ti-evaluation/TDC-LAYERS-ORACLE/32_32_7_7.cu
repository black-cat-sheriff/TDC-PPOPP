#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 1
#define TC 8
#define C 32
#define N 32
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[432];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[6];
  float kernel_shared_local[6];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)))] = (((((1 <= ((((((int)threadIdx.x) * 8) % 27) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 8) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 8) % 9))) && (((((int)threadIdx.x) * 8) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + (((((int)threadIdx.x) * 8) / 27) * 49)) + ((((((int)threadIdx.x) * 8) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 8) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 1) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 1) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 1) % 9))) && ((((((int)threadIdx.x) * 8) + 1) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 1) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 1) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 1) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 2) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 2) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 2) % 9))) && ((((((int)threadIdx.x) * 8) + 2) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 2) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 2) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 2) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 3) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 3) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 3) % 9))) && ((((((int)threadIdx.x) * 8) + 3) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 3) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 3) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 3) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 4) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 4) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 4) % 9))) && ((((((int)threadIdx.x) * 8) + 4) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 4) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 4) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 4) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 5) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 5) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 5) % 9))) && ((((((int)threadIdx.x) * 8) + 5) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 5) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 5) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 5) % 9)) - 8))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 8) + 6) / 27)) < 16) {
      if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 6) / 9)) < 48) {
        if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 426) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 6) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 6) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 6) % 9))) && ((((((int)threadIdx.x) * 8) + 6) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 6) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 6) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 6) % 9)) - 8))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 8) + 7) / 27)) < 16) {
      if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 7) / 9)) < 48) {
        if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 425) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 7) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 7) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 7) % 9))) && ((((((int)threadIdx.x) * 8) + 7) % 9) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 7) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 7) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 7) % 9)) - 8))] : 0.000000e+00f);
          }
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 1))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 2))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 4))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 5))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 7))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 8))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 10))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 11))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 13))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 14))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 15))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 16))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 16))];
    kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 17))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 17))];
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 1134) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 18))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 1133) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 19))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 19))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 7) + 6) / 48) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 7) / 3)) < 126) {
        if (((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 7)) < 378) {
          if (((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) < 1132) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 21)) + 20))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (rc_outer * 144)) + (((int)threadIdx.x) * 21)) + 20))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 144))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 15))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 16))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 21))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 24))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 30))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 33))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 19))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 22))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 25))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 28))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 31))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 34))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 20))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 23))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 29))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 32))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 39))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 42))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 48))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 51))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 37))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 40))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 43))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 46))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 49))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 52))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 38))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 41))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 47))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 50))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 57))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 60))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 66))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 55))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 58))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 61))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 64))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 67))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 56))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 59))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 65))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 68))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 216))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 225))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 234))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 243))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 252))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 261))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 75))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 78))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 81))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 84))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 87))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 226))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 235))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 244))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 253))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 262))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 73))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 76))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 79))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 82))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 85))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 88))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 218))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 227))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 236))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 245))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 254))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 263))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 74))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 77))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 80))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 83))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 86))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 89))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 270))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 279))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 288))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 297))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 306))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 315))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 90))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 93))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 96))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 99))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 102))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 105))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 271))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 280))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 289))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 298))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 307))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 316))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 91))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 94))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 97))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 100))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 103))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 106))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 272))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 281))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 290))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 299))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 308))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 317))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 92))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 95))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 98))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 101))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 104))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 107))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 324))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 333))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 342))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 351))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 360))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 369))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 108))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 111))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 114))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 117))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 120))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 123))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 325))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 334))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 343))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 352))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 361))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 370))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 109))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 112))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 115))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 118))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 121))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 124))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 326))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 335))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 344))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 353))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 362))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 371))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 110))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 113))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 116))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 119))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 122))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 125))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 387))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 396))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 405))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 414))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 423))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 126))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 129))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 132))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 135))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 138))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 141))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 379))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 388))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 397))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 406))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 415))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 424))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 127))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 130))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 133))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 136))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 139))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 142))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 380))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 389))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 398))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 407))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 416))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 425))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + 128))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 144) + 131))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 144) + 134))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 144) + 137))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 144) + 140))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 144) + 143))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,7,4);

        dim3 block(7,1,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


