#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 2
#define TC 16
#define C 192
#define N 160
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[196];
  __shared__ float kernel_shared[20];
  float pad_temp_shared_local[1];
  float kernel_shared_local[1];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 48; ++rc_outer) {
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = (((7 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49)) && (1 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7))) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) - 8))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = ((7 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49)) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) - 7))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = (((7 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49)) && (((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7) < 6)) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) - 6))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = ((1 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7)) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) - 1))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 3))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = data[(((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))];
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 4))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = ((((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7) < 6) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) + 1))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 5))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = (((((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49) < 42) && (1 <= ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7))) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) + 6))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 6))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = ((((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49) < 42) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) + 7))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 7))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    __syncthreads();
    if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) < 196) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 40) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)))] = (((((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 49) < 42) && (((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) % 7) < 6)) ? data[((((((rc_outer * 196) + (((int)threadIdx.z) * 40)) + (((int)threadIdx.y) * 6)) + ((int)threadIdx.x)) + 8))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) >> 2) + ((int)threadIdx.z)) < 5) {
      if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 20) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
          if (((int)threadIdx.x) < 1) {
            kernel_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = kernel[(((((((((int)blockIdx.z) * 8640) + (((int)threadIdx.z) * 1728)) + (rc_outer * 36)) + (((int)threadIdx.x) * 9)) + (((int)threadIdx.y) * 9)) + 8))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 4))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 1))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 2))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 4) + 3))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  }
  compute[(((((((int)blockIdx.z) * 245) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}




class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,1,32);

        dim3 block(7,7,5);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


