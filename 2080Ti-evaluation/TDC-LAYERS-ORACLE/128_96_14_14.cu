#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 4
#define TC 16
#define C 128
#define N 96
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[864];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 1))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 2))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 3))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 3) & 15))) && ((((((int)threadIdx.x) * 19) + 3) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 4))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 4) & 15))) && ((((((int)threadIdx.x) * 19) + 4) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 5))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 5) & 15))) && ((((((int)threadIdx.x) * 19) + 5) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 6))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 6) & 15))) && ((((((int)threadIdx.x) * 19) + 6) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 7))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 7) & 15))) && ((((((int)threadIdx.x) * 19) + 7) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 8))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 8) & 15))) && ((((((int)threadIdx.x) * 19) + 8) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 8) & 15)) - 15))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 759) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 9))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 9) & 15))) && ((((((int)threadIdx.x) * 19) + 9) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 9) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 758) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 10))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 10) & 15))) && ((((((int)threadIdx.x) * 19) + 10) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 10) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 757) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 11))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 11) & 15))) && ((((((int)threadIdx.x) * 19) + 11) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 11) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 756) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 12))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 12) & 15))) && ((((((int)threadIdx.x) * 19) + 12) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 12) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 755) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 13))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 13) & 15))) && ((((((int)threadIdx.x) * 19) + 13) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 13) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 754) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 14))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 14) & 15))) && ((((((int)threadIdx.x) * 19) + 14) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 14) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 753) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 15))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 15) & 15))) && ((((((int)threadIdx.x) * 19) + 15) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 15) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) < 47) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 752) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 16))] = (((((1 <= (((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3))) && ((((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) / 3) * 196)) + (((int)blockIdx.y) * 14)) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 751) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 17))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 750) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 18))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 2))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 4))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 5))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 7))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 8))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 10))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 11))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 13))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 14))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 849) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 848) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 16))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 847) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 17))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 846) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 845) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 19))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 844) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 20))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 96) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 48))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 49))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 50))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 432))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 1))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 433))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 2))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 434))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 9))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 441))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 10))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 442))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 11))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 443))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 64))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 65))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 66))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 3))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 435))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 4))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 436))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 5))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 437))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 12))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 444))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 13))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 445))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 14))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 446))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 80))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 81))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 82))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 6))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 438))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 7))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 439))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 8))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 440))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 15))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 447))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 16))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 448))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 17))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 449))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)) + 588))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,14,16);

        dim3 block(14,1,3);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


