#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 4
#define TC 32
#define C 64
#define N 32
#define H 56
#define W 56

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[7];
  __shared__ float pad_temp_shared[928];
  __shared__ float kernel_shared[288];
  float pad_temp_shared_local[54];
  float kernel_shared_local[18];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)))) && (((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + ((((int)threadIdx.y) * 15) & 3)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) & 3)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) & 3)) - 57))] : 0.000000e+00f);
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 925) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 113) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) < 231) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 924) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 112) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58)) && ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)))) && (((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) / 58) * 3136)) + ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58) * 56)) + (((int)blockIdx.x) * 2)) + ((((int)threadIdx.y) * 15) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 923) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 111) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 922) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 110) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 921) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 109) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) & 3)) - 57))] : 0.000000e+00f);
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + ((((int)threadIdx.y) * 5) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 288) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 36) {
              if ((((((int)blockIdx.z) * 8) + ((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                kernel_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + (((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) % 12) * 3)) + ((((int)threadIdx.y) * 5) % 3)))];
              }
            }
          }
        }
      }
    }
    if (((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + (((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + (((((int)threadIdx.y) * 5) + 1) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 287) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 35) {
              if ((((((int)blockIdx.z) * 8) + (((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                kernel_shared[(((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + 1))] = kernel[(((((((((int)blockIdx.z) * 4608) + ((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + ((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) % 12) * 3)) + (((((int)threadIdx.y) * 5) + 1) % 3)))];
              }
            }
          }
        }
      }
    }
    if (((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + (((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + (((((int)threadIdx.y) * 5) + 2) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 286) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 34) {
              if (((int)threadIdx.x) < 1) {
                if ((((((int)blockIdx.z) * 8) + (((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                  kernel_shared[(((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + 2))] = kernel[(((((((((int)blockIdx.z) * 4608) + ((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + ((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) % 12) * 3)) + (((((int)threadIdx.y) * 5) + 2) % 3)))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 5))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 6))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 8))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 12))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 13))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 16))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 17))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 21))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 22))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 24))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 25))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 26))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 28))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30))];
    pad_temp_shared_local[(24)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32))];
    pad_temp_shared_local[(25)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 33))];
    pad_temp_shared_local[(26)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 34))];
    pad_temp_shared_local[(27)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 232))];
    pad_temp_shared_local[(28)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 233))];
    pad_temp_shared_local[(29)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 234))];
    pad_temp_shared_local[(30)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 236))];
    pad_temp_shared_local[(31)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 237))];
    pad_temp_shared_local[(32)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 238))];
    pad_temp_shared_local[(33)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 240))];
    pad_temp_shared_local[(34)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 241))];
    pad_temp_shared_local[(35)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 242))];
    pad_temp_shared_local[(36)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 244))];
    pad_temp_shared_local[(37)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 245))];
    pad_temp_shared_local[(38)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 246))];
    pad_temp_shared_local[(39)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 248))];
    pad_temp_shared_local[(40)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 249))];
    pad_temp_shared_local[(41)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 250))];
    pad_temp_shared_local[(42)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(43)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 253))];
    pad_temp_shared_local[(44)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 254))];
    pad_temp_shared_local[(45)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 256))];
    pad_temp_shared_local[(46)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 257))];
    pad_temp_shared_local[(47)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 258))];
    pad_temp_shared_local[(48)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 260))];
    pad_temp_shared_local[(49)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 261))];
    pad_temp_shared_local[(50)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 262))];
    pad_temp_shared_local[(51)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 264))];
    pad_temp_shared_local[(52)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 265))];
    pad_temp_shared_local[(53)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 266))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 36) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 36) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 36) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 36) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 36) + 5))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 36) + 6))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 36) + 7))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 36) + 8))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 36) + 9))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 36) + 10))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 36) + 11))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 36) + 12))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 36) + 13))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 36) + 14))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 36) + 15))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 36) + 16))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 36) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(4)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(5)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(6)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(6)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(7)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(7)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(8)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(8)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(9)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(9)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(9)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(9)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(10)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(10)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(10)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(10)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(11)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(11)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(11)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(11)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(12)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(12)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(12)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(12)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(12)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(13)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(13)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(13)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(13)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(13)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(14)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(14)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(14)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(14)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(14)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(15)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(15)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(15)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(15)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(15)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(16)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(16)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(16)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(16)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(16)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(17)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(17)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(17)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(17)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(17)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 464))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 465))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 466))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 468))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 469))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 470))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 472))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 473))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 474))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 476))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 477))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 478))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 484))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 485))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 486))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 488))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 489))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 490))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 492))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 493))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 494))];
    pad_temp_shared_local[(24)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 496))];
    pad_temp_shared_local[(25)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 497))];
    pad_temp_shared_local[(26)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 498))];
    pad_temp_shared_local[(27)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 696))];
    pad_temp_shared_local[(28)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 697))];
    pad_temp_shared_local[(29)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 698))];
    pad_temp_shared_local[(30)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(31)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 701))];
    pad_temp_shared_local[(32)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 702))];
    pad_temp_shared_local[(33)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 704))];
    pad_temp_shared_local[(34)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 705))];
    pad_temp_shared_local[(35)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 706))];
    pad_temp_shared_local[(36)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 708))];
    pad_temp_shared_local[(37)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 709))];
    pad_temp_shared_local[(38)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 710))];
    pad_temp_shared_local[(39)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 712))];
    pad_temp_shared_local[(40)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 713))];
    pad_temp_shared_local[(41)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 714))];
    pad_temp_shared_local[(42)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 716))];
    pad_temp_shared_local[(43)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 717))];
    pad_temp_shared_local[(44)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 718))];
    pad_temp_shared_local[(45)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 720))];
    pad_temp_shared_local[(46)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 721))];
    pad_temp_shared_local[(47)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 722))];
    pad_temp_shared_local[(48)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 724))];
    pad_temp_shared_local[(49)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 725))];
    pad_temp_shared_local[(50)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 726))];
    pad_temp_shared_local[(51)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 728))];
    pad_temp_shared_local[(52)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 729))];
    pad_temp_shared_local[(53)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 730))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 36) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 36) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 36) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 36) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 36) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 36) + 23))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 36) + 24))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 36) + 25))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 36) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 36) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 36) + 28))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 36) + 29))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 36) + 30))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 36) + 31))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 36) + 32))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 36) + 33))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 36) + 34))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 36) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(4)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(5)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(6)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(6)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(7)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(7)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(8)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(8)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(9)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(9)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(9)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(9)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(10)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(10)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(10)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(10)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(11)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(11)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(11)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(11)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(12)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(12)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(12)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(12)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(12)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(13)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(13)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(13)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(13)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(13)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(14)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(14)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(14)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(14)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(14)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(15)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(15)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(15)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(15)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(15)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(16)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(16)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(16)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(16)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(16)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(17)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(17)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(17)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(17)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(17)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(17)]));
  }
  compute[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 56))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 112))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 168))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 224))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 280))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 336))] = compute_local[(6)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(28,1,4);

        dim3 block(2,8,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


