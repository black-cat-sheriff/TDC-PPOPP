#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 5
#define TW 4
#define TC 16
#define C 160
#define N 96
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[640];
  __shared__ float kernel_shared[480];
  float pad_temp_shared_local[16];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 1))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 2))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) - 29))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 3))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) - 29))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 636) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 76) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 4))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 635) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 75) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 5))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) & 1))) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 480) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 60) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 1) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 1) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 479) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 59) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 1))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 1) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 2) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 2) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 478) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 58) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 2))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 2) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 2) % 3)))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) + 1) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 159) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 477) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 57) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 3))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)) + 9))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 4) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 4) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 476) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 56) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 4))] = kernel[(((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 4) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 3))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 33))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 34))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 35))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 64))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 65))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 66))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 67))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 99))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 5))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 6))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 7))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 8))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 9))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 10))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 129))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 130))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 131))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 160))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 161))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 162))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 163))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 195))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 225))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 226))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 227))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 19))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 20))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 21))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 22))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 256))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 257))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 258))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 259))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 291))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 321))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 322))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 323))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 352))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 353))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 354))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 355))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 29))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 30))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 31))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 32))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 33))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 34))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 386))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 387))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 416))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 417))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 418))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 419))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 448))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 449))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 450))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 451))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 483))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 43))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 44))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 45))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 46))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 512))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 513))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 514))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 515))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 544))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 545))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 546))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 547))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 576))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 577))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 578))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 579))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 608))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 609))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 610))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 611))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 53))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 54))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 55))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 56))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 57))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 58))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)))] = (((1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) - 1))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 1))] = (((1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) - 1))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 2))] = (((1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) - 1))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 3))] = (((1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) - 1))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 636) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 76) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 4))] = (((1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) - 1))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 635) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 75) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 5))] = (((1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) - 1))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 480) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 60) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)) + 3))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 1) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 1) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 479) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 59) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 1))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 1) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)) + 3))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 2) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 2) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 478) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 58) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 2))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 2) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 2) % 3)) + 3))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) + 1) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 159) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 477) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 57) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 3))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)) + 12))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 4) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 4) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 476) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 56) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 4))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 4) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)) + 3))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 3))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 33))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 34))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 35))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 64))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 65))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 66))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 67))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 99))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 5))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 6))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 7))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 8))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 9))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 10))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 129))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 130))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 131))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 160))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 161))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 162))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 163))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 195))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 225))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 226))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 227))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 19))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 20))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 21))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 22))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 256))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 257))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 258))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 259))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 291))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 321))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 322))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 323))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 352))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 353))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 354))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 355))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 29))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 30))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 31))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 32))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 33))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 34))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 386))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 387))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 416))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 417))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 418))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 419))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 448))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 449))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 450))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 451))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 483))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 43))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 44))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 45))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 46))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 512))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 513))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 514))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 515))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 544))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 545))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 546))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 547))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 576))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 577))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 578))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 579))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 608))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 609))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 610))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 611))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 53))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 54))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 55))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 56))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 57))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 58))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) & 15)) + 27))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 1))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 1) & 15)) + 27))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 2))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 2) & 15)) + 27))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 3))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 3) & 15)) + 27))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 636) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 76) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 4))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 4) & 15)) + 27))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) < 40) {
      if ((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) < 635) {
        if (((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) < 75) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[(((((((int)threadIdx.z) * 80) + (((int)threadIdx.y) * 40)) + (((int)threadIdx.x) * 6)) + 5))] = ((((((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) & 1)) < 27) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) < 29)) ? data[((((((((rc_outer * 15680) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) >> 1) * 784)) + (((int)blockIdx.y) * 56)) + ((((((int)threadIdx.z) * 5) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) >> 4)) & 1) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.y) * 40) + (((int)threadIdx.x) * 6)) + 5) & 15)) + 27))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 480) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 60) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)) + 6))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 1) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 1) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 479) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 59) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 1))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 1) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)) + 6))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 2) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 2) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 478) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 58) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 2))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 2) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 2) % 3)) + 6))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 10) + ((((int)threadIdx.x) * 5) / 3)) + 1) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + ((((int)threadIdx.x) * 5) / 3)) < 159) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 477) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 57) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 3))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + (((((int)threadIdx.x) * 5) / 3) * 9)) + ((((int)threadIdx.x) * 5) % 3)) + 15))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 10) + (((((int)threadIdx.x) * 5) + 4) / 3)) / 20) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 20) + (((int)threadIdx.y) * 10)) + (((((int)threadIdx.x) * 5) + 4) / 3)) < 160) {
        if ((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) < 476) {
          if (((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 5)) < 56) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[(((((((int)threadIdx.z) * 60) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 5)) + 4))] = kernel[((((((((((int)blockIdx.z) * 11520) + (((int)threadIdx.z) * 1440)) + (rc_outer * 180)) + (((int)threadIdx.y) * 90)) + ((((((int)threadIdx.x) * 5) + 4) / 3) * 9)) + (((((int)threadIdx.x) * 5) + 1) % 3)) + 6))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 3))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 32))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 33))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 34))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 35))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 64))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 65))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 66))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 67))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 96))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 97))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 98))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 99))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 5))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 6))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 7))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 8))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 9))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 10))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 128))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 129))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 130))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 131))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 160))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 161))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 162))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 163))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 192))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 193))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 194))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 195))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 224))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 225))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 226))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 227))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 19))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 20))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 21))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 22))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 256))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 257))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 258))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 259))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 288))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 289))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 290))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 291))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 320))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 321))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 322))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 323))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 352))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 353))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 354))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 355))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 29))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 30))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 31))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 32))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 33))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 34))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 384))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 386))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 387))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 416))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 417))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 418))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 419))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 448))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 449))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 450))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 451))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 483))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 41))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 42))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 43))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 44))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 45))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 46))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 512))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 513))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 514))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 515))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 544))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 545))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 546))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 547))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 576))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 577))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 578))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 579))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 608))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 609))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 610))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 2)) + 611))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 60) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 60) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 60) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 60) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 60) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 60) + 53))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 60) + 54))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 60) + 55))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 60) + 56))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 60) + 57))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 60) + 58))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 60) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(11)]));
  }
  compute[(((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 56)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 3: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 3; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 3; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 3; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 3; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 4: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 4; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 4; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 4; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 4; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 5: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 5; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 5; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 5; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 5; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 4 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[12] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[13] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[14] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[15] += shared_input[c*(TH+2)*(WPAD) + 5 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[16] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[17] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[18] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[19] += shared_input[c*(TH+2)*(WPAD) + 6 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,14,12);

        dim3 block(7,2,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


