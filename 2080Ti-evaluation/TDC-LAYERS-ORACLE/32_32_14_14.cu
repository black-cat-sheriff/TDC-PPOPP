#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 2
#define TC 4
#define C 32
#define N 32
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[576];
  __shared__ float kernel_shared[72];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 576) {
      pad_temp_shared[(((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)))] = (((((9 <= (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144)) && ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) < 15)) ? data[(((((((rc_outer * 784) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) / 144) * 196)) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 575) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 574) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 573) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 572) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 571) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 24) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 72) {
        if (((int)threadIdx.x) < 6) {
          kernel_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))] = kernel[((((((((int)blockIdx.z) * 576) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) / 12) * 288)) + (rc_outer * 36)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) % 12) * 3)) + (((int)threadIdx.x) % 3)))];
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 9) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 144))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 153))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 162))];
    kernel_shared_local[(0)] = kernel_shared[(0)];
    kernel_shared_local[(1)] = kernel_shared[(3)];
    kernel_shared_local[(2)] = kernel_shared[(6)];
    kernel_shared_local[(3)] = kernel_shared[(9)];
    kernel_shared_local[(4)] = kernel_shared[(12)];
    kernel_shared_local[(5)] = kernel_shared[(15)];
    kernel_shared_local[(6)] = kernel_shared[(36)];
    kernel_shared_local[(7)] = kernel_shared[(39)];
    kernel_shared_local[(8)] = kernel_shared[(42)];
    kernel_shared_local[(9)] = kernel_shared[(45)];
    kernel_shared_local[(10)] = kernel_shared[(48)];
    kernel_shared_local[(11)] = kernel_shared[(51)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 145))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 154))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 163))];
    kernel_shared_local[(0)] = kernel_shared[(1)];
    kernel_shared_local[(1)] = kernel_shared[(4)];
    kernel_shared_local[(2)] = kernel_shared[(7)];
    kernel_shared_local[(3)] = kernel_shared[(10)];
    kernel_shared_local[(4)] = kernel_shared[(13)];
    kernel_shared_local[(5)] = kernel_shared[(16)];
    kernel_shared_local[(6)] = kernel_shared[(37)];
    kernel_shared_local[(7)] = kernel_shared[(40)];
    kernel_shared_local[(8)] = kernel_shared[(43)];
    kernel_shared_local[(9)] = kernel_shared[(46)];
    kernel_shared_local[(10)] = kernel_shared[(49)];
    kernel_shared_local[(11)] = kernel_shared[(52)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 146))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 155))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 164))];
    kernel_shared_local[(0)] = kernel_shared[(2)];
    kernel_shared_local[(1)] = kernel_shared[(5)];
    kernel_shared_local[(2)] = kernel_shared[(8)];
    kernel_shared_local[(3)] = kernel_shared[(11)];
    kernel_shared_local[(4)] = kernel_shared[(14)];
    kernel_shared_local[(5)] = kernel_shared[(17)];
    kernel_shared_local[(6)] = kernel_shared[(38)];
    kernel_shared_local[(7)] = kernel_shared[(41)];
    kernel_shared_local[(8)] = kernel_shared[(44)];
    kernel_shared_local[(9)] = kernel_shared[(47)];
    kernel_shared_local[(10)] = kernel_shared[(50)];
    kernel_shared_local[(11)] = kernel_shared[(53)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 288))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 297))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 306))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 432))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 450))];
    kernel_shared_local[(0)] = kernel_shared[(18)];
    kernel_shared_local[(1)] = kernel_shared[(21)];
    kernel_shared_local[(2)] = kernel_shared[(24)];
    kernel_shared_local[(3)] = kernel_shared[(27)];
    kernel_shared_local[(4)] = kernel_shared[(30)];
    kernel_shared_local[(5)] = kernel_shared[(33)];
    kernel_shared_local[(6)] = kernel_shared[(54)];
    kernel_shared_local[(7)] = kernel_shared[(57)];
    kernel_shared_local[(8)] = kernel_shared[(60)];
    kernel_shared_local[(9)] = kernel_shared[(63)];
    kernel_shared_local[(10)] = kernel_shared[(66)];
    kernel_shared_local[(11)] = kernel_shared[(69)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 289))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 298))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 307))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 433))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 442))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 451))];
    kernel_shared_local[(0)] = kernel_shared[(19)];
    kernel_shared_local[(1)] = kernel_shared[(22)];
    kernel_shared_local[(2)] = kernel_shared[(25)];
    kernel_shared_local[(3)] = kernel_shared[(28)];
    kernel_shared_local[(4)] = kernel_shared[(31)];
    kernel_shared_local[(5)] = kernel_shared[(34)];
    kernel_shared_local[(6)] = kernel_shared[(55)];
    kernel_shared_local[(7)] = kernel_shared[(58)];
    kernel_shared_local[(8)] = kernel_shared[(61)];
    kernel_shared_local[(9)] = kernel_shared[(64)];
    kernel_shared_local[(10)] = kernel_shared[(67)];
    kernel_shared_local[(11)] = kernel_shared[(70)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 290))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 299))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 308))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 434))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 443))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 452))];
    kernel_shared_local[(0)] = kernel_shared[(20)];
    kernel_shared_local[(1)] = kernel_shared[(23)];
    kernel_shared_local[(2)] = kernel_shared[(26)];
    kernel_shared_local[(3)] = kernel_shared[(29)];
    kernel_shared_local[(4)] = kernel_shared[(32)];
    kernel_shared_local[(5)] = kernel_shared[(35)];
    kernel_shared_local[(6)] = kernel_shared[(56)];
    kernel_shared_local[(7)] = kernel_shared[(59)];
    kernel_shared_local[(8)] = kernel_shared[(62)];
    kernel_shared_local[(9)] = kernel_shared[(65)];
    kernel_shared_local[(10)] = kernel_shared[(68)];
    kernel_shared_local[(11)] = kernel_shared[(71)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*2+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(0-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(1-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(2-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*2+(3-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(0-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(1-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(2-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*2+(3-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(0-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(1-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(2-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*2+(3-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,1,16);

        dim3 block(7,14,1);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


