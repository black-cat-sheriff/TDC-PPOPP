#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 3
#define TC 4
#define C 32
#define N 32
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[576];
  __shared__ float kernel_shared[72];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 576) {
      pad_temp_shared[(((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)))] = (((((9 <= (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144)) && ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) < 15)) ? data[(((((((rc_outer * 784) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) / 144) * 196)) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 575) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 1) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 574) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 2) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 573) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 3) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 572) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 4) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) < 571) {
      pad_temp_shared[((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5))] = (((((9 <= ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144)) && (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) < 135)) && (1 <= ((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)))) && (((((int)blockIdx.x) * 7) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) < 15)) ? data[(((((((rc_outer * 784) + (((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) / 144) * 196)) + ((((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 144) / 9) * 14)) + (((int)blockIdx.x) * 7)) + ((((((int)threadIdx.y) * 42) + (((int)threadIdx.x) * 6)) + 5) % 9)) - 15))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) < 24) {
      if (((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) < 72) {
        if (((int)threadIdx.x) < 6) {
          kernel_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))] = kernel[((((((((int)blockIdx.z) * 576) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) / 12) * 288)) + (rc_outer * 36)) + ((((((int)threadIdx.y) * 2) + (((int)threadIdx.x) / 3)) % 12) * 3)) + (((int)threadIdx.x) % 3)))];
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 9) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 144))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 153))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 162))];
    kernel_shared_local[(0)] = kernel_shared[(0)];
    kernel_shared_local[(1)] = kernel_shared[(3)];
    kernel_shared_local[(2)] = kernel_shared[(6)];
    kernel_shared_local[(3)] = kernel_shared[(9)];
    kernel_shared_local[(4)] = kernel_shared[(12)];
    kernel_shared_local[(5)] = kernel_shared[(15)];
    kernel_shared_local[(6)] = kernel_shared[(36)];
    kernel_shared_local[(7)] = kernel_shared[(39)];
    kernel_shared_local[(8)] = kernel_shared[(42)];
    kernel_shared_local[(9)] = kernel_shared[(45)];
    kernel_shared_local[(10)] = kernel_shared[(48)];
    kernel_shared_local[(11)] = kernel_shared[(51)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 19))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 145))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 154))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 163))];
    kernel_shared_local[(0)] = kernel_shared[(1)];
    kernel_shared_local[(1)] = kernel_shared[(4)];
    kernel_shared_local[(2)] = kernel_shared[(7)];
    kernel_shared_local[(3)] = kernel_shared[(10)];
    kernel_shared_local[(4)] = kernel_shared[(13)];
    kernel_shared_local[(5)] = kernel_shared[(16)];
    kernel_shared_local[(6)] = kernel_shared[(37)];
    kernel_shared_local[(7)] = kernel_shared[(40)];
    kernel_shared_local[(8)] = kernel_shared[(43)];
    kernel_shared_local[(9)] = kernel_shared[(46)];
    kernel_shared_local[(10)] = kernel_shared[(49)];
    kernel_shared_local[(11)] = kernel_shared[(52)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 11))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 146))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 155))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 164))];
    kernel_shared_local[(0)] = kernel_shared[(2)];
    kernel_shared_local[(1)] = kernel_shared[(5)];
    kernel_shared_local[(2)] = kernel_shared[(8)];
    kernel_shared_local[(3)] = kernel_shared[(11)];
    kernel_shared_local[(4)] = kernel_shared[(14)];
    kernel_shared_local[(5)] = kernel_shared[(17)];
    kernel_shared_local[(6)] = kernel_shared[(38)];
    kernel_shared_local[(7)] = kernel_shared[(41)];
    kernel_shared_local[(8)] = kernel_shared[(44)];
    kernel_shared_local[(9)] = kernel_shared[(47)];
    kernel_shared_local[(10)] = kernel_shared[(50)];
    kernel_shared_local[(11)] = kernel_shared[(53)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 288))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 297))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 306))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 432))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 450))];
    kernel_shared_local[(0)] = kernel_shared[(18)];
    kernel_shared_local[(1)] = kernel_shared[(21)];
    kernel_shared_local[(2)] = kernel_shared[(24)];
    kernel_shared_local[(3)] = kernel_shared[(27)];
    kernel_shared_local[(4)] = kernel_shared[(30)];
    kernel_shared_local[(5)] = kernel_shared[(33)];
    kernel_shared_local[(6)] = kernel_shared[(54)];
    kernel_shared_local[(7)] = kernel_shared[(57)];
    kernel_shared_local[(8)] = kernel_shared[(60)];
    kernel_shared_local[(9)] = kernel_shared[(63)];
    kernel_shared_local[(10)] = kernel_shared[(66)];
    kernel_shared_local[(11)] = kernel_shared[(69)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 289))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 298))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 307))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 433))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 442))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 451))];
    kernel_shared_local[(0)] = kernel_shared[(19)];
    kernel_shared_local[(1)] = kernel_shared[(22)];
    kernel_shared_local[(2)] = kernel_shared[(25)];
    kernel_shared_local[(3)] = kernel_shared[(28)];
    kernel_shared_local[(4)] = kernel_shared[(31)];
    kernel_shared_local[(5)] = kernel_shared[(34)];
    kernel_shared_local[(6)] = kernel_shared[(55)];
    kernel_shared_local[(7)] = kernel_shared[(58)];
    kernel_shared_local[(8)] = kernel_shared[(61)];
    kernel_shared_local[(9)] = kernel_shared[(64)];
    kernel_shared_local[(10)] = kernel_shared[(67)];
    kernel_shared_local[(11)] = kernel_shared[(70)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 290))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 299))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 308))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 434))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 443))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) + 452))];
    kernel_shared_local[(0)] = kernel_shared[(20)];
    kernel_shared_local[(1)] = kernel_shared[(23)];
    kernel_shared_local[(2)] = kernel_shared[(26)];
    kernel_shared_local[(3)] = kernel_shared[(29)];
    kernel_shared_local[(4)] = kernel_shared[(32)];
    kernel_shared_local[(5)] = kernel_shared[(35)];
    kernel_shared_local[(6)] = kernel_shared[(56)];
    kernel_shared_local[(7)] = kernel_shared[(59)];
    kernel_shared_local[(8)] = kernel_shared[(62)];
    kernel_shared_local[(9)] = kernel_shared[(65)];
    kernel_shared_local[(10)] = kernel_shared[(68)];
    kernel_shared_local[(11)] = kernel_shared[(71)];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
  }
  compute[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,1,16);

        dim3 block(7,14,1);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


